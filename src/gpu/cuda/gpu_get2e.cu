#include "hip/hip_runtime.h"
/*
 *  gpu_get2e.cpp
 *  new_quick
 *
 *  Created by Yipu Miao on 6/17/11.
 *  Copyright 2011 University of Florida.All rights reserved.
 *  
 *  Yipu Miao 9/15/11:  the first draft is released. And the GPUGP QM compuation can 
 *                      achieve as much as 15x faster at double precision level compared with CPU.
 */

#include "gpu.h"
#include <hip/hip_runtime.h>

//#define USE_TEXTURE

#if defined(USE_TEXTURE)
  #define USE_TEXTURE_CUTMATRIX
  #define USE_TEXTURE_YCUTOFF
  #define USE_TEXTURE_XCOEFF
#endif

#ifdef USE_TEXTURE_CUTMATRIX
texture <int2, hipTextureType1D, hipReadModeElementType> tex_cutMatrix;
#endif
#ifdef USE_TEXTURE_YCUTOFF
texture <int2, hipTextureType1D, hipReadModeElementType> tex_YCutoff;
#endif
#ifdef USE_TEXTURE_XCOEFF
texture <int2, hipTextureType1D, hipReadModeElementType> tex_Xcoeff;
#endif

//#define USE_ERI_GRAD_STOREADD
//#ifdef USE_ERI_GRAD_STOREADD
//  #define STORE_OPERATOR +=
//#else
//  #define STORE_OPERATOR =  
//#endif

#include "../gpu_get2e_subs_hrr.h"
#if defined(COMPILE_GPU_AOINT)
  #include "../gpu_eri_vertical_int.h"
#endif

#define int_sp
#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_sp.h"
#include "../gpu_get2e_subs.h"
#include "../gpu_eri_grad_assembler_sp.h"
#include "../gpu_get2e_subs_grad.h"

#undef int_sp
#define int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spd.h"
#include "../gpu_get2e_subs.h"
#include "../gpu_eri_grad_assembler_spd.h"
#include "../gpu_get2e_subs_grad.h"


//===================================

#undef int_spd
#define int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_grad_vrr_dddd_1.h"
#include "../gpu_get2e_subs_grad.h"

#undef int_spd
#undef int_spdf
#define int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_grad_vrr_dddd_2.h"
#include "../gpu_get2e_subs_grad.h"


/*
#undef int_spd
#undef int_spdf
#undef int_spdf2
#define int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs_grad.h"
*/

#ifdef GPU_SPDF
//===================================

#undef int_spd
#define int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_1.h"
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#define int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_2.h"
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#define int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_3.h"
#include "../gpu_get2e_subs.h"

#include "../gpu_eri_assembler_spdf_1_2.h"
#include "../gpu_eri_assembler_spdf_2_2.h"
#include "../gpu_eri_assembler_spdf_3_2.h"
#include "../gpu_eri_assembler_spdf_4_2.h"
#include "../gpu_eri_assembler_spdf_5_2.h"
#include "../gpu_eri_assembler_spdf_6_2.h"
#include "../gpu_eri_assembler_spdf_7_2.h"
#include "../gpu_eri_assembler_spdf_8_2.h"

#include "../gpu_eri_grad_assembler_spd_2.h"
#include "../gpu_eri_grad_assembler_spdf_1.h" 
#include "../gpu_eri_grad_assembler_spdf_2.h"
#include "../gpu_eri_grad_assembler_spdf_3.h"
#include "../gpu_eri_grad_assembler_spdf_4.h"
#include "../gpu_eri_grad_assembler_spdf_5.h"
#include "../gpu_eri_grad_assembler_spdf_6.h"
//#include "../gpu_eri_grad_assembler_spdf_7_1.h"
//#include "../gpu_eri_grad_assembler_spdf_7_2.h"
//#include "../gpu_eri_grad_assembler_spdf_7_3.h"
#include "../gpu_get2e_subs_grad.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#define int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_4.h"
//#include "../gpu_eri_grad_vrr_ffff.h"
#include "../gpu_get2e_subs.h"
//#include "../gpu_get2e_subs_grad.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#define int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_5.h"
#include "../gpu_get2e_subs.h"


#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#define int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_6.h"
#include "../gpu_get2e_subs.h"


#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#define int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_7.h"
#include "../gpu_get2e_subs.h"


#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#define int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_8.h"
#include "../gpu_get2e_subs.h"


#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#define int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#define int_spdf10
#include "../gpu_get2e_subs.h"

#endif

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10

//Include the kernels for open shell eri calculations
#define OSHELL

#define int_sp
#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#undef new_quick_2_gpu_get2e_subs_h
#include "../gpu_get2e_subs.h"
#include "../gpu_get2e_subs_grad.h"

#undef int_sp
#define int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"
#include "../gpu_get2e_subs_grad.h"

//===================================

#undef int_spd
#define int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs_grad.h"

#undef int_spd
#undef int_spdf
#define int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs_grad.h"

/*
#undef int_spd
#undef int_spdf
#undef int_spdf2
#define int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs_grad.h"
*/

#ifdef GPU_SPDF
//===================================

#undef int_spd
#define int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#define int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#define int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#define int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#define int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#define int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#define int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#define int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"


#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#define int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#define int_spdf10
#include "../gpu_get2e_subs.h"

#endif

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10

#undef OSHELL


#if defined(USE_TEXTURE)
static void bind_eri_texture(_gpu_type gpu)
{
  #if defined(USE_TEXTURE_CUTMATRIX)
    hipBindTexture(NULL, tex_cutMatrix, gpu->gpu_sim.cutMatrix, sizeof(QUICKDouble) * gpu->nshell * gpu->nshell);
  #endif
  #if defined(USE_TEXTURE_YCUTOFF)
    hipBindTexture(NULL, tex_YCutoff, gpu->gpu_sim.YCutoff, sizeof(QUICKDouble) * gpu->nshell * gpu->nshell);
  #endif
  #if defined(USE_TEXTURE_XCOEFF)
    hipBindTexture(NULL, tex_Xcoeff, gpu->gpu_sim.Xcoeff, sizeof(QUICKDouble) * 4 * gpu->jbasis * gpu->jbasis);
  #endif
}


static void unbind_eri_texture()
{
  #if defined(USE_TEXTURE_CUTMATRIX)
    hipUnbindTexture(tex_cutMatrix);
  #endif
  #if defined(USE_TEXTURE_YCUTOFF)
    hipUnbindTexture(tex_YCutoff);
  #endif
  #if defined(USE_TEXTURE_XCOEFF)
    hipUnbindTexture(tex_Xcoeff);    
  #endif
}
#endif


// totTime is the timer for GPU 2e time. Only on under debug mode
#if defined(DEBUG) || defined(DEBUGTIME)
static float totTime;
#endif


#ifdef COMPILE_GPU_AOINT
// =======   INTERFACE SECTION ===========================
// interface to call Kernel subroutine
void getAOInt(_gpu_type gpu, QUICKULL intStart, QUICKULL intEnd, hipStream_t streamI, int streamID, ERI_entry* aoint_buffer)
{
    QUICK_SAFE_CALL((getAOInt_kernel <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
#ifdef GPU_SPDF
    // Part f-1
    QUICK_SAFE_CALL((getAOInt_kernel_spdf <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-2
    QUICK_SAFE_CALL((getAOInt_kernel_spdf2 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-3
    QUICK_SAFE_CALL((getAOInt_kernel_spdf3 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-4
    QUICK_SAFE_CALL((getAOInt_kernel_spdf4 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-5
    QUICK_SAFE_CALL((getAOInt_kernel_spdf5 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-6
    QUICK_SAFE_CALL((getAOInt_kernel_spdf6 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-7
    QUICK_SAFE_CALL((getAOInt_kernel_spdf7 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-8
    QUICK_SAFE_CALL((getAOInt_kernel_spdf8 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-9
    QUICK_SAFE_CALL((getAOInt_kernel_spdf9 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-10
    QUICK_SAFE_CALL((getAOInt_kernel_spdf10 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
#endif
}
#endif


// interface to call Kernel subroutine
void get2e(_gpu_type gpu)
{
    // Part spd
//    nvtxRangePushA("SCF 2e");

#if defined(USE_TEXTURE)
    bind_eri_texture(gpu);
#endif

    QUICK_SAFE_CALL((k_eri_cshell_sp <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));

    QUICK_SAFE_CALL((k_eri_cshell_spd <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
 
#ifdef GPU_SPDF
    if (gpu->maxL >= 3) {
        // Part f-1
        QUICK_SAFE_CALL((k_eri_cshell_spdf <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-2
        QUICK_SAFE_CALL((k_eri_cshell_spdf2 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-3
        QUICK_SAFE_CALL((k_eri_cshell_spdf3 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-4
        QUICK_SAFE_CALL((k_eri_cshell_spdf4 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-5
        QUICK_SAFE_CALL((k_eri_cshell_spdf5 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-6
        QUICK_SAFE_CALL((k_eri_cshell_spdf6 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-7
        QUICK_SAFE_CALL((k_eri_cshell_spdf7 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-8
        QUICK_SAFE_CALL((k_eri_cshell_spdf8 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-9
//        QUICK_SAFE_CALL((k_eri_cshell_spdf9 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
//                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
//                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
//                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
//                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
//                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
//                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
//                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
//                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
//#if defined(USE_LEGACY_ATOMICS)
//                 gpu->gpu_sim.oULL,
//#else
//                 gpu->gpu_sim.o,
//#endif
//                 gpu->gpu_sim.dense,
//                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
//                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
//                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
//                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
//                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
//#if defined(MPIV_GPU)
//                 gpu->gpu_sim.mpi_bcompute,
//#endif
//                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-10
//        QUICK_SAFE_CALL((k_eri_cshell_spdf10 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
//                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
//                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
//                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
//                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
//                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
//                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
//                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
//                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
//#if defined(USE_LEGACY_ATOMICS)
//                 gpu->gpu_sim.oULL,
//#else
//                 gpu->gpu_sim.o,
//#endif
//                 gpu->gpu_sim.dense,
//                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
//                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
//                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
//                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
//                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
//#if defined(MPIV_GPU)
//                 gpu->gpu_sim.mpi_bcompute,
//#endif
//                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
    }
#endif 

#if defined(USE_TEXTURE)
    unbind_eri_texture();
#endif

//    nvtxRangePop();
}


// interface to call Kernel subroutine for uscf
void get_oshell_eri(_gpu_type gpu)
{
    // Part spd
//    nvtxRangePushA("SCF 2e");

#if defined(USE_TEXTURE)
    bind_eri_texture(gpu);
#endif

    QUICK_SAFE_CALL((k_eri_oshell_sp <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>> 
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));

    QUICK_SAFE_CALL((k_eri_oshell_spd <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>> 
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));

#ifdef GPU_SPDF
    if (gpu->maxL >= 3) {
        // Part f-1
        QUICK_SAFE_CALL((k_eri_oshell_spdf <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>> 
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));

        // Part f-2
        QUICK_SAFE_CALL((k_eri_oshell_spdf2 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-3
        QUICK_SAFE_CALL((k_eri_oshell_spdf3 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-4
        QUICK_SAFE_CALL((k_eri_oshell_spdf4 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-5
        QUICK_SAFE_CALL((k_eri_oshell_spdf5 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-6
        QUICK_SAFE_CALL((k_eri_oshell_spdf6 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-7
        QUICK_SAFE_CALL((k_eri_oshell_spdf7 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-8
        QUICK_SAFE_CALL((k_eri_oshell_spdf8 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-9
//        QUICK_SAFE_CALL((k_eri_oshell_spdf9 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
//                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)>>>
//                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
//                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
//                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
//                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
//                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
//                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
//                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
//#if defined(USE_LEGACY_ATOMICS)
//                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
//#else
//                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
//#endif
//                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
//                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
//                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
//                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
//                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
//                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
//#if defined(MPIV_GPU)
//                 gpu->gpu_sim.mpi_bcompute,
//#endif
//                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        // Part f-10
//        QUICK_SAFE_CALL((k_eri_oshell_spdf10 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
//                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u + 3 * gpu->nbasis)>>>
//                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
//                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
//                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
//                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
//                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
//                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
//                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
//#if defined(USE_LEGACY_ATOMICS)
//                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
//#else
//                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
//#endif
//                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
//                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
//                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
//                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
//                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
//                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
//#if defined(MPIV_GPU)
//                 gpu->gpu_sim.mpi_bcompute,
//#endif
//                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
    }
#endif

#if defined(USE_TEXTURE)
    unbind_eri_texture();
#endif

//    nvtxRangePop();
}


#ifdef COMPILE_GPU_AOINT
// interface to call Kernel subroutine
void getAddInt(_gpu_type gpu, uint32_t bufferSize, ERI_entry* aoint_buffer)
{
    QUICK_SAFE_CALL((k_get_add_int <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> 
                (bufferSize, aoint_buffer, gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.nbasis,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
  #if defined(OSHELL)
                 gpu->gpu_sim.obULL,
  #endif
#else
                 gpu->gpu_sim.o,
  #if defined(OSHELL)
                 gpu->gpu_sim.ob,
  #endif
#endif
                 gpu->gpu_sim.dense
#if defined(OSHELL)
                 , gpu->gpu_sim.denseb
#endif
                 )));
}
#endif


// interface to call Kernel subroutine
void getGrad(_gpu_type gpu)
{
//   nvtxRangePushA("Gradient 2e");
    QUICK_SAFE_CALL((k_get_grad_cshell_sp <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)
#if defined(USE_LEGACY_ATOMICS)
                + sizeof(QUICKULL) * 3u * gpu->natom>>>
#else
                + sizeof(QUICKDouble) * 3u * gpu->natom>>>
#endif
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                 gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                 gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                 gpu->gpu_sim.dense, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                 gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.gradULL,
#else
                 gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.store2,
                 gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC,
                 gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));

    QUICK_SAFE_CALL((k_get_grad_cshell_spd <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)
#if defined(USE_LEGACY_ATOMICS)
                + sizeof(QUICKULL) * 3u * gpu->natom>>>
#else
                + sizeof(QUICKDouble) * 3u * gpu->natom>>>
#endif
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                 gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                 gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                 gpu->gpu_sim.dense, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                 gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.gradULL,
#else
                 gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.store2,
                 gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC,
                 gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));

    // compute one electron gradients in the meantime
    //get_oneen_grad_();

    if (gpu->maxL >= 2) {
        // Part f-1
        QUICK_SAFE_CALL((k_get_grad_cshell_spdf <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                    sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)
#if defined(USE_LEGACY_ATOMICS)
                    + sizeof(QUICKULL) * 3u * gpu->natom>>>
#else
                    + sizeof(QUICKDouble) * 3u * gpu->natom>>>
#endif
                    (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                     gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                     gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                     gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                     gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                     gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                     gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                     gpu->gpu_sim.dense, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                     gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                     gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                     gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                     gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                     gpu->gpu_sim.gradULL,
#else
                     gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                     gpu->gpu_sim.mpi_bcompute,
#endif
                     gpu->gpu_sim.store, gpu->gpu_sim.store2,
                     gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC,
                     gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
        
        // Part f-2
        QUICK_SAFE_CALL((k_get_grad_cshell_spdf2 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                    sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)
#if defined(USE_LEGACY_ATOMICS)
                    + sizeof(QUICKULL) * 3u * gpu->natom>>>
#else
                    + sizeof(QUICKDouble) * 3u * gpu->natom>>>
#endif
                    (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                     gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                     gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                     gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                     gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                     gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                     gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                     gpu->gpu_sim.dense, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                     gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                     gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                     gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                     gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                     gpu->gpu_sim.gradULL,
#else
                     gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                     gpu->gpu_sim.mpi_bcompute,
#endif
                     gpu->gpu_sim.store, gpu->gpu_sim.store2,
                     gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC,
                     gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));

        if (gpu->maxL >= 3) {
#ifdef GPU_SPDF
            // Part f-3
            QUICK_SAFE_CALL((k_get_grad_cshell_spdf3 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                        sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)
#if defined(USE_LEGACY_ATOMICS)
                        + sizeof(QUICKULL) * 3u * gpu->natom>>>
#else
                        + sizeof(QUICKDouble) * 3u * gpu->natom>>>
#endif
                        (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                         gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                         gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                         gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                         gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                         gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                         gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                         gpu->gpu_sim.dense, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                         gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                         gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                         gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                         gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                         gpu->gpu_sim.gradULL,
#else
                         gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                         gpu->gpu_sim.mpi_bcompute,
#endif
                         gpu->gpu_sim.store, gpu->gpu_sim.store2,
                         gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC,
                         gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));

//            QUICK_SAFE_CALL((k_get_grad_cshell_spdf4 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
//                        sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)
//#if defined(USE_LEGACY_ATOMICS)
//                        + sizeof(QUICKULL) * 3u * gpu->natom>>>
//#else
//                        + sizeof(QUICKDouble) * 3u * gpu->natom>>>
//#endif
//                        (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
//                         gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
//                         gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
//                         gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
//                         gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
//                         gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
//                         gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
//                         gpu->gpu_sim.dense, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
//                         gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
//                         gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
//                         gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
//                         gpu->gpu_sim.gradCutoff,
//#if defined(USE_LEGACY_ATOMICS)
//                         gpu->gpu_sim.gradULL,
//#else
//                         gpu->gpu_sim.grad,
//#endif
//#if defined(MPIV_GPU)
//                         gpu->gpu_sim.mpi_bcompute,
//#endif
//                         gpu->gpu_sim.store, gpu->gpu_sim.store2,
//                         gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC,
//                         gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
#endif
        }
    }

//    nvtxRangePop();
}


// interface to call uscf gradient Kernels
void get_oshell_eri_grad(_gpu_type gpu)
{
//    nvtxRangePushA("Gradient 2e");
    QUICK_SAFE_CALL((k_get_grad_oshell_sp <<<gpu->blocks, gpu->gradThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)
#if defined(USE_LEGACY_ATOMICS)
                + sizeof(QUICKULL) * 3u * gpu->natom>>>
#else
                + sizeof(QUICKDouble) * 3u * gpu->natom>>>
#endif
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                 gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                 gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                 gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.gradULL,
#else
                 gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.store2,
                 gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC,
                 gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));

    QUICK_SAFE_CALL((k_get_grad_oshell_spd <<<gpu->blocks, gpu->gradThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)
#if defined(USE_LEGACY_ATOMICS)
                + sizeof(QUICKULL) * 3u * gpu->natom>>>
#else
                + sizeof(QUICKDouble) * 3u * gpu->natom>>>
#endif
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                 gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                 gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                 gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.gradULL,
#else
                 gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.store2,
                 gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC,
                 gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));

    // compute one electron gradients in the meantime
    //get_oneen_grad_();

    if (gpu->maxL >= 2) {
//#ifdef GPU_SPDF
        // Part f-1
        QUICK_SAFE_CALL((k_get_grad_oshell_spdf <<<gpu->blocks, gpu->gradThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)
#if defined(USE_LEGACY_ATOMICS)
                + sizeof(QUICKULL) * 3u * gpu->natom>>>
#else
                + sizeof(QUICKDouble) * 3u * gpu->natom>>>
#endif
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                 gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                 gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                 gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.gradULL,
#else
                 gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.store2,
                 gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC,
                 gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));

        // Part f-2
        QUICK_SAFE_CALL((k_get_grad_oshell_spdf2 <<<gpu->blocks, gpu->gradThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)
#if defined(USE_LEGACY_ATOMICS)
                + sizeof(QUICKULL) * 3u * gpu->natom>>>
#else
                + sizeof(QUICKDouble) * 3u * gpu->natom>>>
#endif
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                 gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                 gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                 gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.gradULL,
#else
                 gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.store2,
                 gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC,
                 gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));

        // Part f-3
	if (gpu->maxL >= 3) {
//            QUICK_SAFE_CALL((k_get_grad_oshell_spdf3 <<<gpu->blocks, gpu->gradThreadsPerBlock,
//                    sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u)
//#if defined(USE_LEGACY_ATOMICS)
//                    + sizeof(QUICKULL) * 3u * gpu->natom>>>
//#else
//                    + sizeof(QUICKDouble) * 3u * gpu->natom>>>
//#endif
//                    (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
//                     gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
//                     gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
//                     gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
//                     gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
//                     gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
//                     gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
//                     gpu->gpu_sim.dense, gpu->gpu_sim.denseb, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
//                     gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
//                     gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
//                     gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
//                     gpu->gpu_sim.gradCutoff,
//#if defined(USE_LEGACY_ATOMICS)
//                     gpu->gpu_sim.gradULL,
//#else
//                     gpu->gpu_sim.grad,
//#endif
//#if defined(MPIV_GPU)
//                     gpu->gpu_sim.mpi_bcompute,
//#endif
//                     gpu->gpu_sim.store, gpu->gpu_sim.store2,
//                     gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC,
//                     gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
//#endif
	}
    }

//    nvtxRangePop();
}


#ifdef COMPILE_GPU_AOINT
// =======   KERNEL SECTION ===========================
__global__ void __launch_bounds__(SM_2X_2E_THREADS_PER_BLOCK, 1) k_get_add_int(uint32_t bufferSize, ERI_entry* aoint_buffer,
        QUICKDouble hyb_coeff, uint32_t nbasis,
#if defined(USE_LEGACY_ATOMICS)
        QUICKULL * const oULL,
  #if defined(OSHELL)
        QUICKULL * const obULL,
  #endif
#else
        QUICKDouble * const o,
  #if defined(OSHELL)
        QUICKDouble * const ob,
  #endif
#endif
        QUICKDouble * const dense
#if defined(OSHELL)
        , QUICKDouble * const denseb
#endif
        )
{
    unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;
    uint32_t const batchSize = 20;
    ERI_entry a[batchSize];
    uint32_t j = 0;
    QUICKDouble temp;
#if defined(OSHELL)
    QUICKDouble temp2;
#endif
 
    QUICKULL myInt = (QUICKULL) (bufferSize) / totalThreads;
    if ((bufferSize - myInt * totalThreads) > offset) myInt++;
    
    for (QUICKULL i = 1; i <= myInt; i++) {
        QUICKULL currentInt = totalThreads * (i - 1) + offset;
        a[j] = aoint_buffer[currentInt];
        j++;

        if (j == batchSize || i == myInt) {
            for (uint32_t k = 0; k < j; k++) {
                uint32_t III = a[k].IJ / nbasis;
                uint32_t JJJ = a[k].IJ % nbasis;
                uint32_t KKK = a[k].KL / nbasis;
                uint32_t LLL = a[k].KL % nbasis;
                
                if (III < nbasis && III >= 0 && JJJ < nbasis && JJJ >= 0
                        && KKK < nbasis && KKK >= 0 && LLL < nbasis && LLL >= 0) {
//                    QUICKDouble hybrid_coeff = 0.0;
//                    if (method == HF) {
//                        hybrid_coeff = 1.0;
//                    } else if (method == B3LYP) {
//                        hybrid_coeff = 0.2;
//                    } else if (method == DFT) {
//                        hybrid_coeff = 0.0;
//                    } else if( method == LIBXC) {
//			hybrid_coeff = hyb_coeff;			
//		    }

#if defined(OSHELL)
                    QUICKDouble DENSELK = (QUICKDouble) (LOC2(dense, LLL, KKK, nbasis, nbasis)
                            + LOC2(denseb, LLL, KKK, nbasis, nbasis));
                    QUICKDouble DENSEJI = (QUICKDouble) (LOC2(dense, JJJ, III, nbasis, nbasis)
                            + LOC2(denseb, JJJ, III, nbasis, nbasis));

                    QUICKDouble DENSEKIA = (QUICKDouble) LOC2(dense, KKK, III, nbasis, nbasis);
                    QUICKDouble DENSEKJA = (QUICKDouble) LOC2(dense, KKK, JJJ, nbasis, nbasis);
                    QUICKDouble DENSELJA = (QUICKDouble) LOC2(dense, LLL, JJJ, nbasis, nbasis);
                    QUICKDouble DENSELIA = (QUICKDouble) LOC2(dense, LLL, III, nbasis, nbasis);

                    QUICKDouble DENSEKIB = (QUICKDouble) LOC2(denseb, KKK, III, nbasis, nbasis);
                    QUICKDouble DENSEKJB = (QUICKDouble) LOC2(denseb, KKK, JJJ, nbasis, nbasis);
                    QUICKDouble DENSELJB = (QUICKDouble) LOC2(denseb, LLL, JJJ, nbasis, nbasis);
                    QUICKDouble DENSELIB = (QUICKDouble) LOC2(denseb, LLL, III, nbasis, nbasis);
#else
                    QUICKDouble DENSEKI = (QUICKDouble) LOC2(dense, KKK, III, nbasis, nbasis);
                    QUICKDouble DENSEKJ = (QUICKDouble) LOC2(dense, KKK, JJJ, nbasis, nbasis);
                    QUICKDouble DENSELJ = (QUICKDouble) LOC2(dense, LLL, JJJ, nbasis, nbasis);
                    QUICKDouble DENSELI = (QUICKDouble) LOC2(dense, LLL, III, nbasis, nbasis);
                    QUICKDouble DENSELK = (QUICKDouble) LOC2(dense, LLL, KKK, nbasis, nbasis);
                    QUICKDouble DENSEJI = (QUICKDouble) LOC2(dense, JJJ, III, nbasis, nbasis);
#endif

                    // ATOMIC ADD VALUE 1
                    temp = (KKK == LLL) ? DENSELK * a[k].value : 2.0 * DENSELK * a[k].value;
                    o_JI += temp;
#if defined(OSHELL)
                    ob_JI += temp;
#endif

                    // ATOMIC ADD VALUE 2
                    if (LLL != JJJ || III != KKK) {
                        temp = (III == JJJ) ? DENSEJI * a[k].value : 2.0 * DENSEJI * a[k].value;
#  if defined(USE_LEGACY_ATOMICS)
                        GPUATOMICADD(&LOC2(oULL, LLL, KKK, nbasis, nbasis), temp, OSCALE);
#  else
                        atomicAdd(&LOC2(o, LLL, KKK, nbasis, nbasis), temp);
#  endif
#if defined(OSHELL)
#  if defined(USE_LEGACY_ATOMICS)
                        GPUATOMICADD(&LOC2(obULL, LLL, KKK, nbasis, nbasis), temp, OSCALE);
#  else
                        atomicAdd(&LOC2(ob, LLL, KKK, nbasis, nbasis), temp);
#  endif
#endif
                    }

                    // ATOMIC ADD VALUE 3
#if defined(OSHELL)
                    temp = (III == KKK && III < JJJ && JJJ < LLL)
                        ? -2.0 * hyb_coeff * DENSELJA * a[k].value : -(hyb_coeff * DENSELJA * a[k].value);
                    temp2 = (III == KKK && III < JJJ && JJJ < LLL)
                        ? -2.0 * hyb_coeff * DENSELJB * a[k].value : -(hyb_coeff * DENSELJB * a[k].value);
                    o_KI += temp;
                    ob_KI += temp2;
#else
                    temp = (III == KKK && III < JJJ && JJJ < LLL)
                        ? -(hyb_coeff * DENSELJ * a[k].value) : -0.5 * hyb_coeff * DENSELJ * a[k].value;
                    o_KI += temp;
#endif

                    // ATOMIC ADD VALUE 4
                    if (KKK != LLL) {
#if defined(OSHELL)
                        temp = -(hyb_coeff * DENSEKJA * a[k].value);
                        temp2 = -(hyb_coeff * DENSEKJB * a[k].value);
#  if defined(USE_LEGACY_ATOMICS)
                        GPUATOMICADD(&LOC2(oULL, LLL, III, nbasis, nbasis), temp, OSCALE);
                        GPUATOMICADD(&LOC2(obULL, LLL, III, nbasis, nbasis), temp2, OSCALE);
#  else
                        atomicAdd(&LOC2(o, LLL, III, nbasis, nbasis), temp);
                        atomicAdd(&LOC2(ob, LLL, III, nbasis, nbasis), temp2);
#  endif
#else
                        temp = -0.5 * hyb_coeff * DENSEKJ * a[k].value;
#  if defined(USE_LEGACY_ATOMICS)
                        GPUATOMICADD(&LOC2(oULL, LLL, III, nbasis, nbasis), temp, OSCALE);
#  else
                        atomicAdd(&LOC2(o, LLL, III, nbasis, nbasis), temp);
#  endif
#endif
                    }

                    // ATOMIC ADD VALUE 5
#if defined(OSHELL)
                    temp = -(hyb_coeff * DENSELIA * a[k].value);
                    temp2 = -(hyb_coeff * DENSELIB * a[k].value);
#else
                    temp = -0.5 * hyb_coeff * DENSELI * a[k].value;
#endif
                    if ((III != JJJ && III < KKK)
                            || (III == JJJ && III == KKK && III < LLL)
                            || (III == KKK && III < JJJ && JJJ < LLL)) {
                        o_JK_MM += temp;
#if defined(OSHELL)
                        ob_JK_MM += temp2;
#endif
                    }

                    // ATOMIC ADD VALUE 5 - 2
                    if (III != JJJ && JJJ == KKK) {
                        o_JK += temp;
#if defined(OSHELL)
                        ob_JK += temp2;
#endif
                    }

                    // ATOMIC ADD VALUE 6
                    if (III != JJJ && KKK != LLL) {
#if defined(OSHELL)
                        temp = -(hyb_coeff * DENSEKIA * a[k].value);
                        temp2 = -(hyb_coeff * DENSEKIB * a[k].value);
#else
                        temp = -0.5 * hyb_coeff * DENSEKI * a[k].value;
#endif
#  if defined(USE_LEGACY_ATOMICS)
                        GPUATOMICADD(&LOC2(oULL, MAX(JJJ, LLL), MIN(JJJ, LLL), nbasis, nbasis), temp, OSCALE);
#  else
                        atomicAdd(&LOC2(o, MAX(JJJ, LLL), MIN(JJJ, LLL), nbasis, nbasis), temp);
#  endif
#if defined(OSHELL)
#  if defined(USE_LEGACY_ATOMICS)
                        GPUATOMICADD(&LOC2(obULL, MAX(JJJ, LLL), MIN(JJJ, LLL), nbasis, nbasis), temp2, OSCALE);
#  else
                        atomicAdd(&LOC2(ob, MAX(JJJ, LLL), MIN(JJJ, LLL), nbasis, nbasis), temp2);
#  endif
#endif

                        // ATOMIC ADD VALUE 6 - 2
                        if (JJJ == LLL && III != KKK) {
#  if defined(USE_LEGACY_ATOMICS)
                            GPUATOMICADD(&LOC2(oULL, LLL, JJJ, nbasis, nbasis), temp, OSCALE);
#  else
                            atomicAdd(&LOC2(o, LLL, JJJ, nbasis, nbasis), temp);
#  endif
#if defined(OSHELL)
#  if defined(USE_LEGACY_ATOMICS)
                            GPUATOMICADD(&LOC2(obULL, LLL, JJJ, nbasis, nbasis), temp2, OSCALE);
#  else
                            atomicAdd(&LOC2(ob, LLL, JJJ, nbasis, nbasis), temp2);
#  endif
#endif
                        }
                    }
                }
            }

            j = 0;
        }
    }
}
#endif
