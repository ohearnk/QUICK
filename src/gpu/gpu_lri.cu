
#include <hip/hip_runtime.h>
/*
  !---------------------------------------------------------------------!
  ! Written by Madu Manathunga on 06/17/2021                            !
  !                                                                     !
  ! Copyright (C) 2020-2021 Merz lab                                    !
  ! Copyright (C) 2020-2021 Götz lab                                    !
  !                                                                     !
  ! This Source Code Form is subject to the terms of the Mozilla Public !
  ! License, v. 2.0. If a copy of the MPL was not distributed with this !
  ! file, You can obtain one at http://mozilla.org/MPL/2.0/.            !
  !_____________________________________________________________________!

  !---------------------------------------------------------------------!
  ! This source file contains functions required for computing 3 center !
  ! integrals necessary for CEW method.                                 !
  !---------------------------------------------------------------------!
*/

#if defined(CEW)

#if defined(CUDA) || defined(CUDA_MPIV)
  #include "cuda/gpu.h"
#elif defined(HIP) || defined(HIP_MPIV)
  #include "hip/gpu.h"
#endif


#include "gpu_lri_subs_hrr.h"

namespace lri {
#include "gpu_lri_vertical_int.h"
}

#define int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_lri_subs.h"
#include "gpu_lri_subs_grad.h"
//===================================
#undef int_spd
#undef int_spdf
#define int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_lri_subs_grad.h"
//===================================
#if defined(GPU_SPDF)
  #undef int_spd
  #undef int_spdf
  #define int_spdf2
  #undef int_spdf3
  #undef int_spdf4
  #undef int_spdf5
  #undef int_spdf6
  #undef int_spdf7
  #undef int_spdf8
  #undef int_spdf9
  #undef int_spdf10
  #include "gpu_lri_subs.h"
#endif
//===================================
#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10


// totTime is the timer for GPU lri time. Only on under debug mode
#if defined(DEBUG) || defined(DEBUGTIME)
static float totTime;
#endif

// =======   INTERFACE SECTION ===========================


// interface to call Kernel subroutine
void get_lri(_gpu_type gpu)
{
    // Part spd
//    nvtxRangePushA("SCF lri");
    QUICK_SAFE_CALL((k_get_lri <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u + 3u * gpu->nbasis)>>>
                (gpu->gpu_sim.natom, gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.allxyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                 gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart, gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis,
                 gpu->gpu_sim.sorted_Qnumber, gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY,
                 gpu->gpu_sim.weightedCenterZ, sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
 
#if defined(GPU_SPDF)
    if (gpu->maxL >= 3) {
        // Part f-2
        QUICK_SAFE_CALL((k_get_lri_spdf2 <<<gpu->blocks, gpu->twoEThreadsPerBlock,
                    sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u + 3u * gpu->nbasis)>>>
                    (gpu->gpu_sim.natom, gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                     gpu->gpu_sim.xyz, gpu->gpu_sim.allxyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                     gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart, gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis,
                     gpu->gpu_sim.sorted_Qnumber, gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                     gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                     gpu->gpu_sim.oULL,
#else
                     gpu->gpu_sim.o,
#endif
                     gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                     gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY,
                     gpu->gpu_sim.weightedCenterZ, sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ,
#if defined(MPIV_GPU)
                     gpu->gpu_sim.mpi_bcompute,
#endif
                     gpu->gpu_sim.store, gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
    }
#endif 

//    nvtxRangePop();
}


// interface to call Kernel subroutine
void get_lri_grad(_gpu_type gpu)
{
//   nvtxRangePushA("Gradient lri");
    QUICK_SAFE_CALL((k_get_lri_grad <<<gpu->blocks, gpu->gradThreadsPerBlock,
                sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u + 3u * gpu->nbasis)
#if defined(USE_LEGACY_ATOMICS)
                + sizeof(QUICKULL) * 3u * gpu->natom>>>
#else
                + sizeof(QUICKDouble) * 3u * gpu->natom>>>
#endif
                (gpu->gpu_sim.natom, gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.allxyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                 gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis,
                 gpu->gpu_sim.sorted_Qnumber, gpu->gpu_sim.sorted_Q,
                 gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start, gpu->gpu_sim.dense,
#if defined(OSHELL)
                 gpu->gpu_sim.denseb,
#endif
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY,
                 gpu->gpu_sim.weightedCenterZ, gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.gradULL,
#else
                 gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.store2, gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB,
                 gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));

    if (gpu->maxL >= 2) {
//#if defined(GPU_SPDF)
        // Part f-2
        QUICK_SAFE_CALL((k_get_lri_grad_spdf2 <<<gpu->blocks, gpu->gradThreadsPerBlock,
                    sizeof(uint32_t) * (TRANSDIM * TRANSDIM * TRANSDIM + 10u + 3u * gpu->nbasis)
#if defined(USE_LEGACY_ATOMICS)
                    + sizeof(QUICKULL) * 3u * gpu->natom>>>
#else
                    + sizeof(QUICKDouble) * 3u * gpu->natom>>>
#endif
                    (gpu->gpu_sim.natom, gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                     gpu->gpu_sim.xyz, gpu->gpu_sim.allxyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                     gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                     gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis,
                     gpu->gpu_sim.sorted_Qnumber, gpu->gpu_sim.sorted_Q,
                     gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo, gpu->gpu_sim.KLMN,
                     gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start, gpu->gpu_sim.dense,
#if defined(OSHELL)
                     gpu->gpu_sim.denseb,
#endif
                     gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                     gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY,
                     gpu->gpu_sim.weightedCenterZ, gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ,
#if defined(USE_LEGACY_ATOMICS)
                     gpu->gpu_sim.gradULL,
#else
                     gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                     gpu->gpu_sim.mpi_bcompute,
#endif
                     gpu->gpu_sim.store, gpu->gpu_sim.store2, gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB,
                     gpu->gpu_sim.trans, gpu->gpu_sim.Sumindex)));
//#endif
    }

//    nvtxRangePop();
}
#endif
