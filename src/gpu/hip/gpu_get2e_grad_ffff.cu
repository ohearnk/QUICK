#include "hip/hip_runtime.h"
/*
 *  gpu_get2e.cpp
 *  new_quick
 *
 *  Created by Yipu Miao on 6/17/11.
 *  Copyright 2011 University of Florida.All rights reserved.
 *
 *  Yipu Miao 9/15/11:  the first draft is released. And the GPUGP QM compuation can
 *                      achieve as much as 15x faster at double precision level compared with CPU.
 */

#include <hip/hip_runtime.h>

#include <iostream>
#include <algorithm>

#include "../gpu_common.h"
#include "gpu_type.h"
#include "gpu_get2e_grad_ffff.h"

/*
 * Constant Memory in GPU is fast but quite limited and hard to operate, usually not allocatable and
 * readonly. So we put the following variables into constant memory:
 * devTrans: arrays to save the mapping index, will be elimited by hand writing unrolling code.
 */
static __constant__ uint8_t devTrans[TRANSDIM * TRANSDIM * TRANSDIM];


//#define USE_TEXTURE
#if defined(USE_TEXTURE)
  #define USE_TEXTURE_CUTMATRIX
  #define USE_TEXTURE_YCUTOFF
  #define USE_TEXTURE_XCOEFF
#endif

#if defined(USE_TEXTURE_CUTMATRIX)
  texture <int2, hipTextureType1D, hipReadModeElementType> tex_cutMatrix;
#endif
#if defined(USE_TEXTURE_YCUTOFF)
  texture <int2, hipTextureType1D, hipReadModeElementType> tex_YCutoff;
#endif
#if defined(USE_TEXTURE_XCOEFF)
  texture <int2, hipTextureType1D, hipReadModeElementType> tex_Xcoeff;
#endif

//#define USE_ERI_GRAD_STOREADD
//#ifdef USE_ERI_GRAD_STOREADD
//  #define STORE_OPERATOR +=
//#else
//  #define STORE_OPERATOR =
//#endif

#define ERI_GRAD_FFFF_TPB (32)
#define ERI_GRAD_FFFF_BPSM (8)

#define ERI_GRAD_FFFF_SMEM_UINT8_SIZE (512)
#define ERI_GRAD_FFFF_SMEM_UINT32_SIZE (5)
#define ERI_GRAD_FFFF_SMEM_UINT32_PTR_SIZE (11)
#define ERI_GRAD_FFFF_SMEM_DBL_SIZE (3)
#define ERI_GRAD_FFFF_SMEM_DBL_PTR_SIZE (18)
#define ERI_GRAD_FFFF_SMEM_CHAR_PTR_SIZE (1)
#define ERI_GRAD_FFFF_SMEM_INT2_PTR_SIZE (1)

#define ERI_GRAD_FFFF_SMEM_PTR_SIZE (1)

#define DEV_SIM_UINT32_PTR_KATOM smem_uint32_ptr[threadIdx.x]
#define DEV_SIM_UINT32_PTR_KPRIM smem_uint32_ptr[ERI_GRAD_FFFF_TPB + threadIdx.x]
#define DEV_SIM_UINT32_PTR_KSTART smem_uint32_ptr[ERI_GRAD_FFFF_TPB * 2 + threadIdx.x]
#define DEV_SIM_UINT32_PTR_KSUMTYPE smem_uint32_ptr[ERI_GRAD_FFFF_TPB * 3 + threadIdx.x]
#define DEV_SIM_UINT32_PTR_PRIM_START smem_uint32_ptr[ERI_GRAD_FFFF_TPB * 4 + threadIdx.x]
#define DEV_SIM_UINT32_PTR_QFBASIS smem_uint32_ptr[ERI_GRAD_FFFF_TPB * 5 + threadIdx.x]
#define DEV_SIM_UINT32_PTR_QSBASIS smem_uint32_ptr[ERI_GRAD_FFFF_TPB * 6 + threadIdx.x]
#define DEV_SIM_UINT32_PTR_QSTART smem_uint32_ptr[ERI_GRAD_FFFF_TPB * 7 + threadIdx.x]
#define DEV_SIM_UINT32_PTR_SORTED_QNUMBER smem_uint32_ptr[ERI_GRAD_FFFF_TPB * 8 + threadIdx.x]
#define DEV_SIM_UINT32_PTR_SORTED_Q smem_uint32_ptr[ERI_GRAD_FFFF_TPB * 9 + threadIdx.x]
#define DEV_SIM_UINT32_PTR_KLMN smem_uint32_ptr[ERI_GRAD_FFFF_TPB + 10 * threadIdx.x]
#define DEV_SIM_INT2_PTR_SORTED_YCUTOFFIJ smem_int2_ptr[threadIdx.x]
#define DEV_SIM_CHAR_PTR_MPI_BCOMPUTE smem_char_ptr[threadIdx.x]
#define DEV_SIM_DBL_PTR_CONS smem_dbl_ptr[threadIdx.x]
#define DEV_SIM_DBL_PTR_CUTMATRIX smem_dbl_ptr[ERI_GRAD_FFFF_TPB + threadIdx.x]
#define DEV_SIM_DBL_PTR_CUTPRIM smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 2 + threadIdx.x]
#define DEV_SIM_DBL_PTR_DENSE smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 3 + threadIdx.x]
#define DEV_SIM_DBL_PTR_DENSEB smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 4 + threadIdx.x]
#define DEV_SIM_DBL_PTR_EXPOSUM smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 5 + threadIdx.x]
#define DEV_SIM_DBL_PTR_GCEXPO smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 6 + threadIdx.x]
#define DEV_SIM_DBL_PTR_STORE smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 7 + threadIdx.x]
#define DEV_SIM_DBL_PTR_STORE2 smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 8 + threadIdx.x]
#define DEV_SIM_DBL_PTR_STOREAA smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 9 + threadIdx.x]
#define DEV_SIM_DBL_PTR_STOREBB smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 10 + threadIdx.x]
#define DEV_SIM_DBL_PTR_STORECC smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 11 + threadIdx.x]
#define DEV_SIM_DBL_PTR_WEIGHTEDCENTERX smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 12 + threadIdx.x]
#define DEV_SIM_DBL_PTR_WEIGHTEDCENTERY smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 13 + threadIdx.x]
#define DEV_SIM_DBL_PTR_WEIGHTEDCENTERZ smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 14 + threadIdx.x]
#define DEV_SIM_DBL_PTR_XCOEFF smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 15 + threadIdx.x]
#define DEV_SIM_DBL_PTR_XYZ smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 16 + threadIdx.x]
#define DEV_SIM_DBL_PTR_YCUTOFF smem_dbl_ptr[ERI_GRAD_FFFF_TPB * 17 + threadIdx.x]
#define DEV_SIM_DBL_PRIMLIMIT smem_dbl[threadIdx.x]
#define DEV_SIM_DBL_GRADCUTOFF smem_dbl[ERI_GRAD_FFFF_TPB + threadIdx.x]
#define DEV_SIM_DBL_HYB_COEFF smem_dbl[ERI_GRAD_FFFF_TPB * 2 + threadIdx.x]
#define DEV_SIM_UINT32_NATOM smem_uint32[threadIdx.x]
#define DEV_SIM_UINT32_NBASIS smem_uint32[ERI_GRAD_FFFF_TPB + threadIdx.x]
#define DEV_SIM_UINT32_NSHELL smem_uint32[ERI_GRAD_FFFF_TPB * 2 + threadIdx.x]
#define DEV_SIM_UINT32_JBASIS smem_uint32[ERI_GRAD_FFFF_TPB * 3 + threadIdx.x]
#define DEV_SIM_UINT32_PRIM_TOTAL smem_uint32[ERI_GRAD_FFFF_TPB * 4 + threadIdx.x]

#define DEV_SIM_PTR_GRAD smem_grad_ptr[threadIdx.x]

#define DEV_SIM_UINT8_TRANS smem_uint8

#if defined(GPU_SPDF)
  #define int_spdf4
  #include "../gpu_eri_grad_vrr_ffff.h"
  #include "gpu_get2e_grad_ffff.cuh"
#endif
#undef int_spdf4

//Include the kernels for open shell eri calculations
#define OSHELL
#if defined(GPU_SPDF)
  #define int_spdf4
//  #include "gpu_get2e_grad_ffff.cuh"
  #endif
#undef OSHELL


// totTime is the timer for GPU 2e time. Only on under debug mode
#if defined(DEBUG) || defined(DEBUGTIME)
  static float totTime;
#endif


struct Partial_ERI {
    int32_t YCutoffIJ_x;
    int32_t YCutoffIJ_y;
    uint32_t Qnumber_x;
    uint32_t Qnumber_y;
    uint32_t kprim_x;
    uint32_t kprim_y;
    uint32_t Q_x;
    uint32_t Q_y;
    uint32_t kprim_score;
};


bool ComparePrimNum(Partial_ERI p1, Partial_ERI p2) {
    return p1.kprim_score > p2.kprim_score;
}


void ResortERIs(_gpu_type gpu) {
    int2 eri_type_order[] = {{0,0}, {0,1}, {1,0}, {1,1},
        {0,2}, {2,0}, {1,2}, {2,1},
        {0,3}, {3,0}, {2,2}, {1,3},
        {3,1}, {2,3}, {3,2}, {3,3}};
    unsigned char eri_type_order_map[] = {0, 1, 3, 6, 10, 13, 15, 16};
    uint32_t eri_type_block_map[17];
    int2 *resorted_YCutoffIJ = (int2 *) malloc(sizeof(int2) * gpu->gpu_cutoff->sqrQshell);
    bool ffset = false;

    // Step 1: sort according sum of angular momentum of a partial ERI. (ie. i+j of <ij| ).
    // Step 2: sort according to type order specified in eri_type_order array. This ensures that eri vector follows the order we
    // want.
    uint32_t idx1 = 0;
    uint32_t idx2 = 0;
    uint32_t ffStart = 0;

    for (uint32_t ij_sum = 0; ij_sum <= 6; ij_sum++) {
        for (uint32_t ieto = eri_type_order_map[ij_sum]; ieto < eri_type_order_map[ij_sum + 1]; ieto++) {
            int2 lbl_t = eri_type_order[ieto];
            eri_type_block_map[idx2] = idx1;
            for (uint32_t i = 0; i < gpu->gpu_cutoff->sqrQshell; i++) {
                if ((int) gpu->gpu_basis->sorted_Qnumber->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].x] == lbl_t.x
                        && (int) gpu->gpu_basis->sorted_Qnumber->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].y] == lbl_t.y) {
                    resorted_YCutoffIJ[idx1].x = gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].x;
                    resorted_YCutoffIJ[idx1].y = gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].y;
                    idx1++;
                }
            }
            idx2++;
        }
    }

    eri_type_block_map[idx2]=idx1;

    for (uint32_t i = 0; i < gpu->gpu_cutoff->sqrQshell; i++) {
        gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].x=resorted_YCutoffIJ[i].x;
        gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].y=resorted_YCutoffIJ[i].y;

        if (ffset == false
                && gpu->gpu_basis->sorted_Qnumber->_hostData[resorted_YCutoffIJ[i].x]
                + gpu->gpu_basis->sorted_Qnumber->_hostData[resorted_YCutoffIJ[i].y] == 6) {
            ffStart = i;
            ffset = true;
        }
    }

    // create an array of structs
    Partial_ERI *partial_eris = (Partial_ERI *) malloc(sizeof(Partial_ERI) * gpu->gpu_cutoff->sqrQshell);

    for (uint32_t i = 0; i < gpu->gpu_cutoff->sqrQshell; i++) {
        uint32_t kprim1
            = gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].x]];
        uint32_t kprim2
            = gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].y]];
        uint32_t kprim_score = 10 * std::max(kprim1, kprim2) + std::min(kprim1, kprim2) + (kprim1 + kprim2);
        partial_eris[i] = {gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].x,
            gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].y,
            gpu->gpu_basis->sorted_Qnumber->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].x],
            gpu->gpu_basis->sorted_Qnumber->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].y],
            gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].x]],
            gpu->gpu_basis->kprim->_hostData[gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].y]],
            gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].x],
            gpu->gpu_basis->sorted_Q->_hostData[gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].y],
            kprim_score};
    }

    for (uint32_t i = 0; i < 16; i++) {
        std::sort(partial_eris + eri_type_block_map[i], partial_eris + eri_type_block_map[i + 1], ComparePrimNum);
    }

    for (uint32_t i = 0; i < gpu->gpu_cutoff->sqrQshell; i++) {
        gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].x = partial_eris[i].YCutoffIJ_x;
        gpu->gpu_cutoff->sorted_YCutoffIJ->_hostData[i].y = partial_eris[i].YCutoffIJ_y;
    }

    gpu->gpu_cutoff->sorted_YCutoffIJ->Upload();
    gpu->gpu_sim.sorted_YCutoffIJ = gpu->gpu_cutoff->sorted_YCutoffIJ->_devData;
    gpu->gpu_sim.ffStart = ffStart;

    free(resorted_YCutoffIJ);
    free(partial_eris);
}


void getGrad_ffff(_gpu_type gpu)
{
    ResortERIs(gpu);

    uint8_t trans[TRANSDIM * TRANSDIM * TRANSDIM] = {};
    uint32_t *uint32_buffer = (uint32_t *) malloc(sizeof(uint32_t) * ERI_GRAD_FFFF_SMEM_UINT32_SIZE * ERI_GRAD_FFFF_TPB);
    uint32_t **uint32_ptr_buffer = (uint32_t **) malloc(sizeof(uint32_t *) * ERI_GRAD_FFFF_SMEM_UINT32_PTR_SIZE * ERI_GRAD_FFFF_TPB);
    QUICKDouble *dbl_buffer = (QUICKDouble*) malloc(sizeof(QUICKDouble) * ERI_GRAD_FFFF_SMEM_DBL_SIZE*ERI_GRAD_FFFF_TPB);
    QUICKDouble **dbl_ptr_buffer = (QUICKDouble **) malloc(sizeof(QUICKDouble *) * ERI_GRAD_FFFF_SMEM_DBL_PTR_SIZE * ERI_GRAD_FFFF_TPB);
    int2 **int2_ptr_buffer = (int2 **) malloc(sizeof(int2 *) * ERI_GRAD_FFFF_SMEM_INT2_PTR_SIZE * ERI_GRAD_FFFF_TPB);
    unsigned char **char_ptr_buffer = (unsigned char **) malloc(sizeof(unsigned char *) * ERI_GRAD_FFFF_SMEM_CHAR_PTR_SIZE * ERI_GRAD_FFFF_TPB);
    QUICKAtomicType **grad_ptr_buffer = (QUICKAtomicType **) malloc(sizeof(QUICKAtomicType *) * ERI_GRAD_FFFF_SMEM_PTR_SIZE * ERI_GRAD_FFFF_TPB);

    for (uint32_t i = 0; i < ERI_GRAD_FFFF_TPB; i++) {
        uint32_buffer[i] = gpu->gpu_sim.natom;
        uint32_buffer[ERI_GRAD_FFFF_TPB + i] = gpu->gpu_sim.nbasis;
        uint32_buffer[ERI_GRAD_FFFF_TPB * 2 + i] = gpu->gpu_sim.nshell;
        uint32_buffer[ERI_GRAD_FFFF_TPB * 3 + i] = gpu->gpu_sim.jbasis;
        uint32_buffer[ERI_GRAD_FFFF_TPB * 4 + i] = gpu->gpu_sim.prim_total;
    }

    for (uint32_t i = 0; i < ERI_GRAD_FFFF_TPB; i++) {
        uint32_ptr_buffer[i] = gpu->gpu_sim.katom;
        uint32_ptr_buffer[ERI_GRAD_FFFF_TPB + i] = gpu->gpu_sim.kprim;
        uint32_ptr_buffer[ERI_GRAD_FFFF_TPB * 2 + i] = gpu->gpu_sim.kstart;
        uint32_ptr_buffer[ERI_GRAD_FFFF_TPB * 3 + i] = gpu->gpu_sim.Ksumtype;
        uint32_ptr_buffer[ERI_GRAD_FFFF_TPB * 4 + i] = gpu->gpu_sim.prim_start;
        uint32_ptr_buffer[ERI_GRAD_FFFF_TPB * 5 + i] = gpu->gpu_sim.Qfbasis;
        uint32_ptr_buffer[ERI_GRAD_FFFF_TPB * 6 + i] = gpu->gpu_sim.Qsbasis;
        uint32_ptr_buffer[ERI_GRAD_FFFF_TPB * 7 + i] = gpu->gpu_sim.Qstart;
        uint32_ptr_buffer[ERI_GRAD_FFFF_TPB * 8 + i] = gpu->gpu_sim.sorted_Q;
        uint32_ptr_buffer[ERI_GRAD_FFFF_TPB * 9 + i] = gpu->gpu_sim.sorted_Qnumber;
        uint32_ptr_buffer[ERI_GRAD_FFFF_TPB * 10 + i] = gpu->gpu_sim.KLMN;
    }

    for (uint32_t i = 0; i < ERI_GRAD_FFFF_TPB; i++) {
        dbl_buffer[i] = gpu->gpu_sim.primLimit;
        dbl_buffer[ERI_GRAD_FFFF_TPB + i] = gpu->gpu_sim.gradCutoff;
        dbl_buffer[ERI_GRAD_FFFF_TPB * 2 + i] = gpu->gpu_sim.hyb_coeff;
    }

    for (uint32_t i = 0; i < ERI_GRAD_FFFF_TPB; i++) {
        dbl_ptr_buffer[i] = gpu->gpu_sim.cons;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB + i] = gpu->gpu_sim.cutMatrix;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 2 + i] = gpu->gpu_sim.cutPrim;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 3 + i] = gpu->gpu_sim.dense;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 4 + i] = gpu->gpu_sim.denseb;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 5 + i] = gpu->gpu_sim.expoSum;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 6 + i] = gpu->gpu_sim.gcexpo;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 7 + i] = gpu->gpu_sim.store;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 8 + i] = gpu->gpu_sim.store2;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 9 + i] = gpu->gpu_sim.storeAA;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 10 + i] = gpu->gpu_sim.storeBB;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 11 + i] = gpu->gpu_sim.storeCC;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 12 + i] = gpu->gpu_sim.weightedCenterX;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 13 + i] = gpu->gpu_sim.weightedCenterY;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 14 + i] = gpu->gpu_sim.weightedCenterZ;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 15 + i] = gpu->gpu_sim.Xcoeff;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 16 + i] = gpu->gpu_sim.xyz;
        dbl_ptr_buffer[ERI_GRAD_FFFF_TPB * 17 + i] = gpu->gpu_sim.YCutoff;
    }

    for (uint32_t i = 0; i < ERI_GRAD_FFFF_TPB; i++) {
        int2_ptr_buffer[i] = gpu->gpu_sim.sorted_YCutoffIJ;
    }

    for (uint32_t i = 0; i < ERI_GRAD_FFFF_TPB; i++) {
        char_ptr_buffer[i] = gpu->gpu_sim.mpi_bcompute;
    }

    for (uint32_t i = 0; i < ERI_GRAD_FFFF_TPB; i++) {
#if defined(USE_LEGACY_ATOMICS)
        grad_ptr_buffer[i] = gpu->gpu_sim.gradULL;
#else
        grad_ptr_buffer[i] = gpu->gpu_sim.grad;
#endif
    }

    LOC3(trans, 0, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 0;
    LOC3(trans, 0, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 3;
    LOC3(trans, 0, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 9;
    LOC3(trans, 0, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 19;
    LOC3(trans, 0, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 34;
    LOC3(trans, 0, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 55;
    LOC3(trans, 0, 0, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 83;
    LOC3(trans, 0, 0, 7, TRANSDIM, TRANSDIM, TRANSDIM) = 119;
    LOC3(trans, 0, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 2;
    LOC3(trans, 0, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 5;
    LOC3(trans, 0, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 16;
    LOC3(trans, 0, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 31;
    LOC3(trans, 0, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 47;
    LOC3(trans, 0, 1, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 66;
    LOC3(trans, 0, 1, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 99;
    LOC3(trans, 0, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 8;
    LOC3(trans, 0, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 15;
    LOC3(trans, 0, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 22;
    LOC3(trans, 0, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 41;
    LOC3(trans, 0, 2, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 72;
    LOC3(trans, 0, 2, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 105;
    LOC3(trans, 0, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 18;
    LOC3(trans, 0, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 30;
    LOC3(trans, 0, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 42;
    LOC3(trans, 0, 3, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 78;
    LOC3(trans, 0, 3, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 111;
    LOC3(trans, 0, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 33;
    LOC3(trans, 0, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 48;
    LOC3(trans, 0, 4, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 73;
    LOC3(trans, 0, 4, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 112;
    LOC3(trans, 0, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 54;
    LOC3(trans, 0, 5, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 67;
    LOC3(trans, 0, 5, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 106;
    LOC3(trans, 0, 6, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 82;
    LOC3(trans, 0, 6, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 100;
    LOC3(trans, 0, 7, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 118;
    LOC3(trans, 1, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 1;
    LOC3(trans, 1, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 6;
    LOC3(trans, 1, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 14;
    LOC3(trans, 1, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 27;
    LOC3(trans, 1, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 49;
    LOC3(trans, 1, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 68;
    LOC3(trans, 1, 0, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 101;
    LOC3(trans, 1, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 4;
    LOC3(trans, 1, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 10;
    LOC3(trans, 1, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 25;
    LOC3(trans, 1, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 40;
    LOC3(trans, 1, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 58;
    LOC3(trans, 1, 1, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 86;
    LOC3(trans, 1, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 12;
    LOC3(trans, 1, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 24;
    LOC3(trans, 1, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 35;
    LOC3(trans, 1, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 59;
    LOC3(trans, 1, 2, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 87;
    LOC3(trans, 1, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 29;
    LOC3(trans, 1, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 39;
    LOC3(trans, 1, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 60;
    LOC3(trans, 1, 3, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 93;
    LOC3(trans, 1, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 51;
    LOC3(trans, 1, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 57;
    LOC3(trans, 1, 4, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 88;
    LOC3(trans, 1, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 70;
    LOC3(trans, 1, 5, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 85;
    LOC3(trans, 1, 6, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 103;
    LOC3(trans, 2, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 7;
    LOC3(trans, 2, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 13;
    LOC3(trans, 2, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 21;
    LOC3(trans, 2, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 43;
    LOC3(trans, 2, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 74;
    LOC3(trans, 2, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 109;
    LOC3(trans, 2, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 11;
    LOC3(trans, 2, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 23;
    LOC3(trans, 2, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 36;
    LOC3(trans, 2, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 61;
    LOC3(trans, 2, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 89;
    LOC3(trans, 2, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 20;
    LOC3(trans, 2, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 37;
    LOC3(trans, 2, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 65;
    LOC3(trans, 2, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 98;
    LOC3(trans, 2, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 45;
    LOC3(trans, 2, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 63;
    LOC3(trans, 2, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 97;
    LOC3(trans, 2, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 76;
    LOC3(trans, 2, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 91;
    LOC3(trans, 2, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 109;
    LOC3(trans, 3, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 17;
    LOC3(trans, 3, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 26;
    LOC3(trans, 3, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 44;
    LOC3(trans, 3, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 79;
    LOC3(trans, 3, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 113;
    LOC3(trans, 3, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 28;
    LOC3(trans, 3, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 38;
    LOC3(trans, 3, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 62;
    LOC3(trans, 3, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 94;
    LOC3(trans, 3, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 46;
    LOC3(trans, 3, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 64;
    LOC3(trans, 3, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 96;
    LOC3(trans, 3, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 80;
    LOC3(trans, 3, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 95;
    LOC3(trans, 3, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 115;
    LOC3(trans, 4, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 32;
    LOC3(trans, 4, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 50;
    LOC3(trans, 4, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 75;
    LOC3(trans, 4, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 114;
    LOC3(trans, 4, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 52;
    LOC3(trans, 4, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 56;
    LOC3(trans, 4, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 90;
    LOC3(trans, 4, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 77;
    LOC3(trans, 4, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 92;
    LOC3(trans, 4, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 116;
    LOC3(trans, 5, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 53;
    LOC3(trans, 5, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 69;
    LOC3(trans, 5, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 108;
    LOC3(trans, 5, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 71;
    LOC3(trans, 5, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 84;
    LOC3(trans, 5, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 110;
    LOC3(trans, 6, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 81;
    LOC3(trans, 6, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 102;
    LOC3(trans, 6, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 104;
    LOC3(trans, 7, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 117;

    uint8_t *dev_uint8_buffer;
    uint32_t *dev_uint32_buffer;
    uint32_t **dev_uint32_ptr_buffer;
    QUICKDouble *dev_dbl_buffer;
    QUICKDouble **dev_dbl_ptr_buffer;
    int2 **dev_int2_ptr_buffer;
    unsigned char **dev_char_ptr_buffer;
    QUICKAtomicType **dev_grad_ptr_buffer;

    gpuMalloc((void **) &dev_uint8_buffer, sizeof(uint8_t) * ERI_GRAD_FFFF_SMEM_UINT8_SIZE);
    gpuMalloc((void **) &dev_uint32_buffer, sizeof(uint32_t) * ERI_GRAD_FFFF_SMEM_UINT32_SIZE * ERI_GRAD_FFFF_TPB);
    gpuMalloc((void **) &dev_uint32_ptr_buffer, sizeof(uint32_t *) * ERI_GRAD_FFFF_SMEM_UINT32_PTR_SIZE * ERI_GRAD_FFFF_TPB);
    gpuMalloc((void **) &dev_dbl_buffer, sizeof(QUICKDouble) * ERI_GRAD_FFFF_SMEM_DBL_SIZE * ERI_GRAD_FFFF_TPB);
    gpuMalloc((void **) &dev_dbl_ptr_buffer, sizeof(QUICKDouble *) * ERI_GRAD_FFFF_SMEM_DBL_PTR_SIZE * ERI_GRAD_FFFF_TPB);
    gpuMalloc((void **) &dev_int2_ptr_buffer, sizeof(int2 *) * ERI_GRAD_FFFF_SMEM_INT2_PTR_SIZE * ERI_GRAD_FFFF_TPB);
    gpuMalloc((void **) &dev_char_ptr_buffer, sizeof(unsigned char *) * ERI_GRAD_FFFF_SMEM_CHAR_PTR_SIZE * ERI_GRAD_FFFF_TPB);
    gpuMalloc((void **) &dev_grad_ptr_buffer, sizeof(QUICKAtomicType *) * ERI_GRAD_FFFF_SMEM_PTR_SIZE * ERI_GRAD_FFFF_TPB);

    gpuMemcpy(dev_uint8_buffer, &trans, sizeof(uint8_t) * ERI_GRAD_FFFF_SMEM_UINT8_SIZE, hipMemcpyHostToDevice);
    gpuMemcpy(dev_uint32_buffer, uint32_buffer, sizeof(uint32_t) * ERI_GRAD_FFFF_SMEM_UINT32_SIZE * ERI_GRAD_FFFF_TPB, hipMemcpyHostToDevice);
    gpuMemcpy(dev_uint32_ptr_buffer, uint32_ptr_buffer, sizeof(uint32_t *) * ERI_GRAD_FFFF_SMEM_UINT32_PTR_SIZE * ERI_GRAD_FFFF_TPB, hipMemcpyHostToDevice);
    gpuMemcpy(dev_dbl_buffer, dbl_buffer, sizeof(QUICKDouble) * ERI_GRAD_FFFF_SMEM_DBL_SIZE * ERI_GRAD_FFFF_TPB, hipMemcpyHostToDevice);
    gpuMemcpy(dev_dbl_ptr_buffer, dbl_ptr_buffer, sizeof(QUICKDouble *) * ERI_GRAD_FFFF_SMEM_DBL_PTR_SIZE * ERI_GRAD_FFFF_TPB, hipMemcpyHostToDevice);
    gpuMemcpy(dev_int2_ptr_buffer, int2_ptr_buffer, sizeof(int2 *) * ERI_GRAD_FFFF_SMEM_INT2_PTR_SIZE * ERI_GRAD_FFFF_TPB, hipMemcpyHostToDevice);
    gpuMemcpy(dev_char_ptr_buffer, char_ptr_buffer, sizeof(unsigned char *) * ERI_GRAD_FFFF_SMEM_CHAR_PTR_SIZE * ERI_GRAD_FFFF_TPB, hipMemcpyHostToDevice);
    gpuMemcpy(dev_grad_ptr_buffer, grad_ptr_buffer, sizeof(QUICKAtomicType *) * ERI_GRAD_FFFF_SMEM_PTR_SIZE * ERI_GRAD_FFFF_TPB, hipMemcpyHostToDevice);

    // Part f-3
    if (gpu->maxL >= 3) {
#ifdef GPU_SPDF
        QUICK_SAFE_CALL((getGrad_kernel_ffff <<<gpu->blocks * ERI_GRAD_FFFF_BPSM, ERI_GRAD_FFFF_TPB,
                    (sizeof(uint8_t) * ERI_GRAD_FFFF_SMEM_UINT8_SIZE
                     + sizeof(uint32_t) * ERI_GRAD_FFFF_SMEM_UINT32_SIZE
                     + sizeof(QUICKDouble) * ERI_GRAD_FFFF_SMEM_DBL_SIZE
                     + sizeof(QUICKDouble *) * ERI_GRAD_FFFF_SMEM_DBL_PTR_SIZE
                     + sizeof(uint32_t *) * ERI_GRAD_FFFF_SMEM_UINT32_PTR_SIZE
                     + sizeof(int2 *) * ERI_GRAD_FFFF_SMEM_INT2_PTR_SIZE
                     + sizeof(unsigned char *) * ERI_GRAD_FFFF_SMEM_CHAR_PTR_SIZE
                     + sizeof(QUICKAtomicType *) * ERI_GRAD_FFFF_SMEM_PTR_SIZE) * ERI_GRAD_FFFF_TPB>>>
                    (dev_uint8_buffer, dev_uint8_ptr_buffer, dev_uint32_buffer, dev_uint32_ptr_buffer, dev_dbl_buffer,
                     dev_dbl_ptr_buffer, dev_int2_ptr_buffer, dev_char_ptr_buffer, dev_grad_ptr_buffer,
                     gpu->gpu_sim.ffStart, gpu->gpu_sim.sqrQshell)));

#endif
    }

    free(uint32_buffer);
    free(uint32_ptr_buffer);
    free(dbl_buffer);
    free(dbl_ptr_buffer);
    free(int2_ptr_buffer);
    free(char_ptr_buffer);
    free(grad_ptr_buffer);

    gpuFree(dev_uint8_buffer);
    gpuFree(dev_uint32_buffer);
    gpuFree(dev_uint32_ptr_buffer);
    gpuFree(dev_dbl_buffer);
    gpuFree(dev_dbl_ptr_buffer);
    gpuFree(dev_int2_ptr_buffer);
    gpuFree(dev_char_ptr_buffer);
    gpuFree(dev_grad_ptr_buffer);
}


// interface to call uscf gradient Kernels
void get_oshell_eri_grad_ffff(_gpu_type gpu)
{
//   nvtxRangePushA("Gradient 2e");

    // compute one electron gradients in the meantime
//    get_oneen_grad_();

    // Part f-3
//    if (gpu->maxL >= 3) {
//        QUICK_SAFE_CALL((getGrad_oshell_kernel_ffff <<<gpu->blocks, gpu->gradThreadsPerBlock>>> ()))
//#endif
//    }

//    nvtxRangePop();
}


void upload_para_to_const_ffff()
{
    uint8_t trans[TRANSDIM * TRANSDIM * TRANSDIM] = {};

    LOC3(trans, 0, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 0;
    LOC3(trans, 0, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 3;
    LOC3(trans, 0, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 9;
    LOC3(trans, 0, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 19;
    LOC3(trans, 0, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 34;
    LOC3(trans, 0, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 55;
    LOC3(trans, 0, 0, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 83;
    LOC3(trans, 0, 0, 7, TRANSDIM, TRANSDIM, TRANSDIM) = 119;
    LOC3(trans, 0, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 2;
    LOC3(trans, 0, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 5;
    LOC3(trans, 0, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 16;
    LOC3(trans, 0, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 31;
    LOC3(trans, 0, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 47;
    LOC3(trans, 0, 1, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 66;
    LOC3(trans, 0, 1, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 99;
    LOC3(trans, 0, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 8;
    LOC3(trans, 0, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 15;
    LOC3(trans, 0, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 22;
    LOC3(trans, 0, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 41;
    LOC3(trans, 0, 2, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 72;
    LOC3(trans, 0, 2, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 105;
    LOC3(trans, 0, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 18;
    LOC3(trans, 0, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 30;
    LOC3(trans, 0, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 42;
    LOC3(trans, 0, 3, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 78;
    LOC3(trans, 0, 3, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 111;
    LOC3(trans, 0, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 33;
    LOC3(trans, 0, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 48;
    LOC3(trans, 0, 4, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 73;
    LOC3(trans, 0, 4, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 112;
    LOC3(trans, 0, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 54;
    LOC3(trans, 0, 5, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 67;
    LOC3(trans, 0, 5, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 106;
    LOC3(trans, 0, 6, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 82;
    LOC3(trans, 0, 6, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 100;
    LOC3(trans, 0, 7, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 118;
    LOC3(trans, 1, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 1;
    LOC3(trans, 1, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 6;
    LOC3(trans, 1, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 14;
    LOC3(trans, 1, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 27;
    LOC3(trans, 1, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 49;
    LOC3(trans, 1, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 68;
    LOC3(trans, 1, 0, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 101;
    LOC3(trans, 1, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 4;
    LOC3(trans, 1, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 10;
    LOC3(trans, 1, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 25;
    LOC3(trans, 1, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 40;
    LOC3(trans, 1, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 58;
    LOC3(trans, 1, 1, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 86;
    LOC3(trans, 1, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 12;
    LOC3(trans, 1, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 24;
    LOC3(trans, 1, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 35;
    LOC3(trans, 1, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 59;
    LOC3(trans, 1, 2, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 87;
    LOC3(trans, 1, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 29;
    LOC3(trans, 1, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 39;
    LOC3(trans, 1, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 60;
    LOC3(trans, 1, 3, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 93;
    LOC3(trans, 1, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 51;
    LOC3(trans, 1, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 57;
    LOC3(trans, 1, 4, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 88;
    LOC3(trans, 1, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 70;
    LOC3(trans, 1, 5, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 85;
    LOC3(trans, 1, 6, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 103;
    LOC3(trans, 2, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 7;
    LOC3(trans, 2, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 13;
    LOC3(trans, 2, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 21;
    LOC3(trans, 2, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 43;
    LOC3(trans, 2, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 74;
    LOC3(trans, 2, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 109;
    LOC3(trans, 2, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 11;
    LOC3(trans, 2, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 23;
    LOC3(trans, 2, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 36;
    LOC3(trans, 2, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 61;
    LOC3(trans, 2, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 89;
    LOC3(trans, 2, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 20;
    LOC3(trans, 2, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 37;
    LOC3(trans, 2, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 65;
    LOC3(trans, 2, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 98;
    LOC3(trans, 2, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 45;
    LOC3(trans, 2, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 63;
    LOC3(trans, 2, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 97;
    LOC3(trans, 2, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 76;
    LOC3(trans, 2, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 91;
    LOC3(trans, 2, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 109;
    LOC3(trans, 3, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 17;
    LOC3(trans, 3, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 26;
    LOC3(trans, 3, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 44;
    LOC3(trans, 3, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 79;
    LOC3(trans, 3, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 113;
    LOC3(trans, 3, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 28;
    LOC3(trans, 3, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 38;
    LOC3(trans, 3, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 62;
    LOC3(trans, 3, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 94;
    LOC3(trans, 3, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 46;
    LOC3(trans, 3, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 64;
    LOC3(trans, 3, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 96;
    LOC3(trans, 3, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 80;
    LOC3(trans, 3, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 95;
    LOC3(trans, 3, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 115;
    LOC3(trans, 4, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 32;
    LOC3(trans, 4, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 50;
    LOC3(trans, 4, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 75;
    LOC3(trans, 4, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 114;
    LOC3(trans, 4, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 52;
    LOC3(trans, 4, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 56;
    LOC3(trans, 4, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 90;
    LOC3(trans, 4, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 77;
    LOC3(trans, 4, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 92;
    LOC3(trans, 4, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 116;
    LOC3(trans, 5, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 53;
    LOC3(trans, 5, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 69;
    LOC3(trans, 5, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 108;
    LOC3(trans, 5, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 71;
    LOC3(trans, 5, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 84;
    LOC3(trans, 5, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 110;
    LOC3(trans, 6, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 81;
    LOC3(trans, 6, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 102;
    LOC3(trans, 6, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 104;
    LOC3(trans, 7, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 117;

    gpuMemcpyToSymbol((const void *) devTrans, (const void *) trans,
            sizeof(uint8_t) * TRANSDIM * TRANSDIM * TRANSDIM);
}
