#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/*
 *  gpu_get2e.cpp
 *  new_quick
 *
 *  Created by Yipu Miao on 6/17/11.
 *  Copyright 2011 University of Florida.All rights reserved.
 *  
 *  Yipu Miao 9/15/11:  the first draft is released. And the GPUGP QM compuation can 
 *                      achieve as much as 15x faster at double precision level compared with CPU.
 */

#include "gpu.h"
#include <hip/hip_runtime.h>

/*
 Constant Memory in GPU is fast but quite limited and hard to operate, usually not allocatable and 
 readonly. So we put the following variables into constant memory:
 devTrans : arrays to save the mapping index, will be elimited by hand writing unrolling code.
 Sumindex: a array to store refect how many temp variable needed in VRR. can be elimited by hand writing code.
 */
static __constant__ uint8_t devTrans[TRANSDIM * TRANSDIM * TRANSDIM];
static __constant__ uint8_t Sumindex[10] = {0, 0, 1, 4, 10, 20, 35, 56, 84, 120};

//#define USE_TEXTURE

#if defined(USE_TEXTURE)
  #define USE_TEXTURE_CUTMATRIX
  #define USE_TEXTURE_YCUTOFF
  #define USE_TEXTURE_XCOEFF
#endif

#ifdef USE_TEXTURE_CUTMATRIX
texture <int2, hipTextureType1D, hipReadModeElementType> tex_cutMatrix;
#endif
#ifdef USE_TEXTURE_YCUTOFF
texture <int2, hipTextureType1D, hipReadModeElementType> tex_YCutoff;
#endif
#ifdef USE_TEXTURE_XCOEFF
texture <int2, hipTextureType1D, hipReadModeElementType> tex_Xcoeff;
#endif

//#define USE_ERI_GRAD_STOREADD
//#ifdef USE_ERI_GRAD_STOREADD
//  #define STORE_OPERATOR +=
//#else
//  #define STORE_OPERATOR =  
//#endif

#include "../gpu_get2e_subs_hrr.h"
#if defined(COMPILE_GPU_AOINT)
  #include "../gpu_eri_vertical_int.h"
#endif

#define int_sp
#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_sp.h"
#include "../gpu_get2e_subs.h"
#include "../gpu_eri_grad_assembler_sp.h"
#include "../gpu_get2e_subs_grad.h"

#undef int_sp
#define int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spd.h"
#include "../gpu_get2e_subs.h"
#include "../gpu_eri_grad_assembler_spd.h"
#include "../gpu_get2e_subs_grad.h"


//===================================

#undef int_spd
#define int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_grad_vrr_dddd_1.h"
#include "../gpu_get2e_subs_grad.h"

#undef int_spd
#undef int_spdf
#define int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_grad_vrr_dddd_2.h"
#include "../gpu_get2e_subs_grad.h"


/*
#undef int_spd
#undef int_spdf
#undef int_spdf2
#define int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs_grad.h"
*/

#ifdef GPU_SPDF
//===================================

#undef int_spd
#define int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_1.h"
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#define int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_2.h"
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#define int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_3.h"
#include "../gpu_get2e_subs.h"

#include "../gpu_eri_assembler_spdf_1_2.h"
#include "../gpu_eri_assembler_spdf_2_2.h"
#include "../gpu_eri_assembler_spdf_3_2.h"
#include "../gpu_eri_assembler_spdf_4_2.h"
#include "../gpu_eri_assembler_spdf_5_2.h"
#include "../gpu_eri_assembler_spdf_6_2.h"
#include "../gpu_eri_assembler_spdf_7_2.h"
#include "../gpu_eri_assembler_spdf_8_2.h"

#include "../gpu_eri_grad_assembler_spd_2.h"
#include "../gpu_eri_grad_assembler_spdf_1.h" 
#include "../gpu_eri_grad_assembler_spdf_2.h"
#include "../gpu_eri_grad_assembler_spdf_3.h"
#include "../gpu_eri_grad_assembler_spdf_4.h"
#include "../gpu_eri_grad_assembler_spdf_5.h"
#include "../gpu_eri_grad_assembler_spdf_6.h"
//#include "../gpu_eri_grad_assembler_spdf_7_1.h"
//#include "../gpu_eri_grad_assembler_spdf_7_2.h"
//#include "../gpu_eri_grad_assembler_spdf_7_3.h"
#include "../gpu_get2e_subs_grad.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#define int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_4.h"
//#include "../gpu_eri_grad_vrr_ffff.h"
#include "../gpu_get2e_subs.h"
//#include "../gpu_get2e_subs_grad.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#define int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_5.h"
#include "../gpu_get2e_subs.h"


#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#define int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_6.h"
#include "../gpu_get2e_subs.h"


#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#define int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_7.h"
#include "../gpu_get2e_subs.h"


#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#define int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_eri_assembler_spdf_8.h"
#include "../gpu_get2e_subs.h"


#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#define int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#define int_spdf10
#include "../gpu_get2e_subs.h"

#endif

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10

//Include the kernels for open shell eri calculations
#define OSHELL

#define int_sp
#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#undef new_quick_2_gpu_get2e_subs_h
#include "../gpu_get2e_subs.h"
#include "../gpu_get2e_subs_grad.h"

#undef int_sp
#define int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"
#include "../gpu_get2e_subs_grad.h"

//===================================

#undef int_spd
#define int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs_grad.h"

#undef int_spd
#undef int_spdf
#define int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs_grad.h"

/*
#undef int_spd
#undef int_spdf
#undef int_spdf2
#define int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs_grad.h"
*/

#ifdef GPU_SPDF
//===================================

#undef int_spd
#define int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#define int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#define int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#define int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#define int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#define int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#define int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#define int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"


#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#define int_spdf9
#undef int_spdf10
#include "../gpu_get2e_subs.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#define int_spdf10
#include "../gpu_get2e_subs.h"

#endif

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10

#undef OSHELL


#if defined(USE_TEXTURE)
static void bind_eri_texture(_gpu_type gpu)
{
  #if defined(USE_TEXTURE_CUTMATRIX)
    hipBindTexture(NULL, tex_cutMatrix, gpu->gpu_sim.cutMatrix, sizeof(QUICKDouble) * gpu->nshell * gpu->nshell);
  #endif
  #if defined(USE_TEXTURE_YCUTOFF)
    hipBindTexture(NULL, tex_YCutoff, gpu->gpu_sim.YCutoff, sizeof(QUICKDouble) * gpu->nshell * gpu->nshell);
  #endif
  #if defined(USE_TEXTURE_XCOEFF)
    hipBindTexture(NULL, tex_Xcoeff, gpu->gpu_sim.Xcoeff, sizeof(QUICKDouble) * 4 * gpu->jbasis * gpu->jbasis);
  #endif
}


static void unbind_eri_texture()
{
  #if defined(USE_TEXTURE_CUTMATRIX)
    hipUnbindTexture(tex_cutMatrix);
  #endif
  #if defined(USE_TEXTURE_YCUTOFF)
    hipUnbindTexture(tex_YCutoff);
  #endif
  #if defined(USE_TEXTURE_XCOEFF)
    hipUnbindTexture(tex_Xcoeff);    
  #endif
}
#endif


// totTime is the timer for GPU 2e time. Only on under debug mode
#if defined(DEBUG) || defined(DEBUGTIME)
static float totTime;
#endif


#ifdef COMPILE_GPU_AOINT
// =======   INTERFACE SECTION ===========================
// interface to call Kernel subroutine
void getAOInt(_gpu_type gpu, QUICKULL intStart, QUICKULL intEnd, hipStream_t streamI, int streamID, ERI_entry* aoint_buffer)
{
    QUICK_SAFE_CALL((getAOInt_kernel <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
#ifdef GPU_SPDF
    // Part f-1
    QUICK_SAFE_CALL((getAOInt_kernel_spdf <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-2
    QUICK_SAFE_CALL((getAOInt_kernel_spdf2 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-3
    QUICK_SAFE_CALL((getAOInt_kernel_spdf3 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-4
    QUICK_SAFE_CALL((getAOInt_kernel_spdf4 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-5
    QUICK_SAFE_CALL((getAOInt_kernel_spdf5 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-6
    QUICK_SAFE_CALL((getAOInt_kernel_spdf6 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-7
    QUICK_SAFE_CALL((getAOInt_kernel_spdf7 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-8
    QUICK_SAFE_CALL((getAOInt_kernel_spdf8 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-9
    QUICK_SAFE_CALL((getAOInt_kernel_spdf9 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
    // Part f-10
    QUICK_SAFE_CALL((getAOInt_kernel_spdf10 <<<gpu->blocks, gpu->twoEThreadsPerBlock, 0, streamI>>> (intStart, intEnd, aoint_buffer, streamID)));
#endif
}
#endif


// interface to call Kernel subroutine
void get2e(_gpu_type gpu)
{
    // Part spd
//    nvtxRangePushA("SCF 2e");

#if defined(USE_TEXTURE)
    bind_eri_texture(gpu);
#endif

    QUICK_SAFE_CALL((k_eri_cshell_sp <<<gpu->blocks, gpu->twoEThreadsPerBlock>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));

    QUICK_SAFE_CALL((k_eri_cshell_spd <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
 
#ifdef GPU_SPDF
    if (gpu->maxL >= 3) {
        // Part f-1
        QUICK_SAFE_CALL((k_eri_cshell_spdf <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
        // Part f-2
        QUICK_SAFE_CALL((k_eri_cshell_spdf2 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
        // Part f-3
        QUICK_SAFE_CALL((k_eri_cshell_spdf3 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
        // Part f-4
        QUICK_SAFE_CALL((k_eri_cshell_spdf4 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
        // Part f-5
        QUICK_SAFE_CALL((k_eri_cshell_spdf5 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
        // Part f-6
        QUICK_SAFE_CALL((k_eri_cshell_spdf6 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
        // Part f-7
        QUICK_SAFE_CALL((k_eri_cshell_spdf7 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
        // Part f-8
        QUICK_SAFE_CALL((k_eri_cshell_spdf8 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
#else
                 gpu->gpu_sim.o,
#endif
                 gpu->gpu_sim.dense,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
        // Part f-9
//        QUICK_SAFE_CALL((k_eri_cshell_spdf9 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
//                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
//                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
//                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
//                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
//                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
//                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
//#if defined(USE_LEGACY_ATOMICS)
//                 gpu->gpu_sim.oULL,
//#else
//                 gpu->gpu_sim.o,
//#endif
//                 gpu->gpu_sim.dense,
//                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
//                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
//                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
//                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
//                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
//#if defined(MPIV_GPU)
//                 gpu->gpu_sim.mpi_bcompute,
//#endif
//                 gpu->gpu_sim.store)));
        // Part f-10
//        QUICK_SAFE_CALL((k_eri_cshell_spdf10 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
//                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
//                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
//                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
//                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
//                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
//                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
//#if defined(USE_LEGACY_ATOMICS)
//                 gpu->gpu_sim.oULL,
//#else
//                 gpu->gpu_sim.o,
//#endif
//                 gpu->gpu_sim.dense,
//                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
//                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
//                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
//                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
//                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
//#if defined(MPIV_GPU)
//                 gpu->gpu_sim.mpi_bcompute,
//#endif
//                 gpu->gpu_sim.store)));
    }
#endif 

#if defined(USE_TEXTURE)
    unbind_eri_texture();
#endif

//    nvtxRangePop();
}


// interface to call Kernel subroutine for uscf
void get_oshell_eri(_gpu_type gpu)
{
    // Part spd
//    nvtxRangePushA("SCF 2e");

#if defined(USE_TEXTURE)
    bind_eri_texture(gpu);
#endif

    QUICK_SAFE_CALL((k_eri_oshell_sp <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> 
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));

    QUICK_SAFE_CALL((k_eri_oshell_spd <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> 
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));

#ifdef GPU_SPDF
    if (gpu->maxL >= 3) {
        // Part f-1
        QUICK_SAFE_CALL((k_eri_oshell_spdf <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> 
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));

        // Part f-2
        QUICK_SAFE_CALL((k_eri_oshell_spdf2 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
        // Part f-3
        QUICK_SAFE_CALL((k_eri_oshell_spdf3 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
        // Part f-4
        QUICK_SAFE_CALL((k_eri_oshell_spdf4 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
        // Part f-5
        QUICK_SAFE_CALL((k_eri_oshell_spdf5 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
        // Part f-6
        QUICK_SAFE_CALL((k_eri_oshell_spdf6 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
        // Part f-7
        QUICK_SAFE_CALL((k_eri_oshell_spdf7 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
        // Part f-8
        QUICK_SAFE_CALL((k_eri_oshell_spdf8 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
#else
                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
#endif
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store)));
        // Part f-9
//        QUICK_SAFE_CALL((k_eri_oshell_spdf9 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>>
//                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
//                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
//                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
//                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
//                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
//                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
//                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
//#if defined(USE_LEGACY_ATOMICS)
//                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
//#else
//                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
//#endif
//                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
//                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
//                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
//                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
//                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
//                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
//#if defined(MPIV_GPU)
//                 gpu->gpu_sim.mpi_bcompute,
//#endif
//                 gpu->gpu_sim.store)));
        // Part f-10
//        QUICK_SAFE_CALL((k_eri_oshell_spdf10 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>>
//                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
//                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
//                 gpu->gpu_sim.xyz, gpu->gpu_sim.fStart, gpu->gpu_sim.ffStart, gpu->gpu_sim.kstart,
//                 gpu->gpu_sim.katom, gpu->gpu_sim.kprim, gpu->gpu_sim.Qstart,
//                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
//                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.KLMN,
//                 gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
//#if defined(USE_LEGACY_ATOMICS)
//                 gpu->gpu_sim.oULL, gpu->gpu_sim.obULL,
//#else
//                 gpu->gpu_sim.o, gpu->gpu_sim.ob,
//#endif
//                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb,
//                 gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
//                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
//                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
//                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.integralCutoff,
//                 gpu->gpu_sim.primLimit, gpu->gpu_sim.maxIntegralCutoff, gpu->gpu_sim.leastIntegralCutoff,
//#if defined(MPIV_GPU)
//                 gpu->gpu_sim.mpi_bcompute,
//#endif
//                 gpu->gpu_sim.store)));
    }
#endif

#if defined(USE_TEXTURE)
    unbind_eri_texture();
#endif

//    nvtxRangePop();
}


#ifdef COMPILE_GPU_AOINT
// interface to call Kernel subroutine
void getAddInt(_gpu_type gpu, uint32_t bufferSize, ERI_entry* aoint_buffer)
{
    QUICK_SAFE_CALL((k_get_add_int <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> 
                (bufferSize, aoint_buffer, gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.nbasis,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.oULL,
  #if defined(OSHELL)
                 gpu->gpu_sim.obULL,
  #endif
#else
                 gpu->gpu_sim.o,
  #if defined(OSHELL)
                 gpu->gpu_sim.ob,
  #endif
#endif
                 gpu->gpu_sim.dense
#if defined(OSHELL)
                 , gpu->gpu_sim.denseb
#endif
                 )));
}
#endif


// interface to call Kernel subroutine
void getGrad(_gpu_type gpu)
{
//   nvtxRangePushA("Gradient 2e");
    QUICK_SAFE_CALL((k_get_grad_cshell_sp <<<gpu->blocks, gpu->twoEThreadsPerBlock>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                 gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                 gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                 gpu->gpu_sim.dense, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                 gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.gradULL,
#else
                 gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.store2,
                 gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC)));

    QUICK_SAFE_CALL((k_get_grad_cshell_spd <<<gpu->blocks, gpu->twoEThreadsPerBlock>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                 gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                 gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                 gpu->gpu_sim.dense, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                 gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.gradULL,
#else
                 gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.store2,
                 gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC)));

    // compute one electron gradients in the meantime
    //get_oneen_grad_();

    if (gpu->maxL >= 2) {
        // Part f-1
        QUICK_SAFE_CALL((k_get_grad_cshell_spdf <<<gpu->blocks, gpu->twoEThreadsPerBlock>>>
                    (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                     gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                     gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                     gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                     gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                     gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                     gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                     gpu->gpu_sim.dense, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                     gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                     gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                     gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                     gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                     gpu->gpu_sim.gradULL,
#else
                     gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                     gpu->gpu_sim.mpi_bcompute,
#endif
                     gpu->gpu_sim.store, gpu->gpu_sim.store2,
                     gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC)));
        
        // Part f-2
        QUICK_SAFE_CALL((k_get_grad_cshell_spdf2 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>>
                    (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                     gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                     gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                     gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                     gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                     gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                     gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                     gpu->gpu_sim.dense, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                     gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                     gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                     gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                     gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                     gpu->gpu_sim.gradULL,
#else
                     gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                     gpu->gpu_sim.mpi_bcompute,
#endif
                     gpu->gpu_sim.store, gpu->gpu_sim.store2,
                     gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC)));

        if (gpu->maxL >= 3) {
#ifdef GPU_SPDF
            // Part f-3
            QUICK_SAFE_CALL((k_get_grad_cshell_spdf3 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>>
                    (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                     gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                     gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                     gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                     gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                     gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                     gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                     gpu->gpu_sim.dense, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                     gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                     gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                     gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                     gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                     gpu->gpu_sim.gradULL,
#else
                     gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                     gpu->gpu_sim.mpi_bcompute,
#endif
                     gpu->gpu_sim.store, gpu->gpu_sim.store2,
                     gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC)));

//            QUICK_SAFE_CALL((k_get_grad_cshell_spdf4 <<<gpu->blocks, gpu->twoEThreadsPerBlock>>> ()))
//                    (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
//                     gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
//                     gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
//                     gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
//                     gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
//                     gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
//                     gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
//                     gpu->gpu_sim.dense, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
//                     gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
//                     gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
//                     gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
//                     gpu->gpu_sim.gradCutoff,
//#if defined(USE_LEGACY_ATOMICS)
//                     gpu->gpu_sim.gradULL,
//#else
//                     gpu->gpu_sim.grad,
//#endif
//#if defined(MPIV_GPU)
//                     gpu->gpu_sim.mpi_bcompute,
//#endif
//                     gpu->gpu_sim.store, gpu->gpu_sim.store2,
//                     gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC)));
#endif
        }
    }

//    nvtxRangePop();
}


// interface to call uscf gradient Kernels
void get_oshell_eri_grad(_gpu_type gpu)
{
//    nvtxRangePushA("Gradient 2e");
    QUICK_SAFE_CALL((k_get_grad_oshell_sp <<<gpu->blocks, gpu->gradThreadsPerBlock>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                 gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                 gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                 gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.gradULL,
#else
                 gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.store2,
                 gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC)));

    QUICK_SAFE_CALL((k_get_grad_oshell_spd <<<gpu->blocks, gpu->gradThreadsPerBlock>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                 gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                 gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                 gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.gradULL,
#else
                 gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.store2,
                 gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC)));

    // compute one electron gradients in the meantime
    //get_oneen_grad_();

    if (gpu->maxL >= 2) {
//#ifdef GPU_SPDF
        // Part f-1
        QUICK_SAFE_CALL((k_get_grad_oshell_spdf <<<gpu->blocks, gpu->gradThreadsPerBlock>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                 gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                 gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                 gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.gradULL,
#else
                 gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.store2,
                 gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC)));

        // Part f-2
        QUICK_SAFE_CALL((k_get_grad_oshell_spdf2 <<<gpu->blocks, gpu->gradThreadsPerBlock>>>
                (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
                 gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
                 gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
                 gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
                 gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
                 gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
                 gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
                 gpu->gpu_sim.dense, gpu->gpu_sim.denseb, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
                 gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
                 gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
                 gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
                 gpu->gpu_sim.gradCutoff,
#if defined(USE_LEGACY_ATOMICS)
                 gpu->gpu_sim.gradULL,
#else
                 gpu->gpu_sim.grad,
#endif
#if defined(MPIV_GPU)
                 gpu->gpu_sim.mpi_bcompute,
#endif
                 gpu->gpu_sim.store, gpu->gpu_sim.store2,
                 gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC)));

        // Part f-3
	if (gpu->maxL >= 3) {
//            QUICK_SAFE_CALL((k_get_grad_oshell_spdf3 <<<gpu->blocks, gpu->gradThreadsPerBlock>>>
//                    (gpu->gpu_sim.hyb_coeff, gpu->gpu_sim.natom,
//                     gpu->gpu_sim.nbasis, gpu->gpu_sim.nshell, gpu->gpu_sim.jbasis,
//                     gpu->gpu_sim.xyz, gpu->gpu_sim.kstart, gpu->gpu_sim.katom,
//                     gpu->gpu_sim.kprim, gpu->gpu_sim.Ksumtype, gpu->gpu_sim.Qstart,
//                     gpu->gpu_sim.Qsbasis, gpu->gpu_sim.Qfbasis, gpu->gpu_sim.sorted_Qnumber,
//                     gpu->gpu_sim.sorted_Q, gpu->gpu_sim.cons, gpu->gpu_sim.gcexpo,
//                     gpu->gpu_sim.KLMN, gpu->gpu_sim.prim_total, gpu->gpu_sim.prim_start,
//                     gpu->gpu_sim.dense, gpu->gpu_sim.denseb, gpu->gpu_sim.Xcoeff, gpu->gpu_sim.expoSum,
//                     gpu->gpu_sim.weightedCenterX, gpu->gpu_sim.weightedCenterY, gpu->gpu_sim.weightedCenterZ,
//                     gpu->gpu_sim.sqrQshell, gpu->gpu_sim.sorted_YCutoffIJ, gpu->gpu_sim.cutMatrix,
//                     gpu->gpu_sim.YCutoff, gpu->gpu_sim.cutPrim, gpu->gpu_sim.primLimit,
//                     gpu->gpu_sim.gradCutoff,
//#if defined(USE_LEGACY_ATOMICS)
//                     gpu->gpu_sim.gradULL,
//#else
//                     gpu->gpu_sim.grad,
//#endif
//#if defined(MPIV_GPU)
//                     gpu->gpu_sim.mpi_bcompute,
//#endif
//                     gpu->gpu_sim.store, gpu->gpu_sim.store2,
//                     gpu->gpu_sim.storeAA, gpu->gpu_sim.storeBB, gpu->gpu_sim.storeCC)));
//#endif
	}
    }

//    nvtxRangePop();
}


#ifdef COMPILE_GPU_AOINT
// =======   KERNEL SECTION ===========================
__global__ void __launch_bounds__(SM_2X_2E_THREADS_PER_BLOCK, 1) k_get_add_int(uint32_t bufferSize, ERI_entry* aoint_buffer,
        QUICKDouble hyb_coeff, uint32_t nbasis,
#if defined(USE_LEGACY_ATOMICS)
        QUICKULL * const oULL,
  #if defined(OSHELL)
        QUICKULL * const obULL,
  #endif
#else
        QUICKDouble * const o,
  #if defined(OSHELL)
        QUICKDouble * const ob,
  #endif
#endif
        QUICKDouble * const dense
#if defined(OSHELL)
        , QUICKDouble * const denseb
#endif
        )
{
    unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;
    uint32_t const batchSize = 20;
    ERI_entry a[batchSize];
    uint32_t j = 0;
    QUICKDouble temp;
#if defined(OSHELL)
    QUICKDouble temp2;
#endif
 
    QUICKULL myInt = (QUICKULL) (bufferSize) / totalThreads;
    if ((bufferSize - myInt * totalThreads) > offset) myInt++;
    
    for (QUICKULL i = 1; i <= myInt; i++) {
        QUICKULL currentInt = totalThreads * (i - 1) + offset;
        a[j] = aoint_buffer[currentInt];
        j++;

        if (j == batchSize || i == myInt) {
            for (uint32_t k = 0; k < j; k++) {
                uint8_t III = a[k].IJ / nbasis;
                uint8_t JJJ = a[k].IJ % nbasis;
                uint8_t KKK = a[k].KL / nbasis;
                uint8_t LLL = a[k].KL % nbasis;
                
                if (III < nbasis && III >= 0 && JJJ < nbasis && JJJ >= 0
                        && KKK < nbasis && KKK >= 0 && LLL < nbasis && LLL >= 0) {
//                    QUICKDouble hybrid_coeff = 0.0;
//                    if (method == HF) {
//                        hybrid_coeff = 1.0;
//                    } else if (method == B3LYP) {
//                        hybrid_coeff = 0.2;
//                    } else if (method == DFT) {
//                        hybrid_coeff = 0.0;
//                    } else if( method == LIBXC) {
//			hybrid_coeff = hyb_coeff;			
//		    }

#if defined(OSHELL)
                    QUICKDouble DENSELK = (QUICKDouble) (LOC2(dense, LLL, KKK, nbasis, nbasis)
                            + LOC2(denseb, LLL, KKK, nbasis, nbasis));
                    QUICKDouble DENSEJI = (QUICKDouble) (LOC2(dense, JJJ, III, nbasis, nbasis)
                            + LOC2(denseb, JJJ, III, nbasis, nbasis));

                    QUICKDouble DENSEKIA = (QUICKDouble) LOC2(dense, KKK, III, nbasis, nbasis);
                    QUICKDouble DENSEKJA = (QUICKDouble) LOC2(dense, KKK, JJJ, nbasis, nbasis);
                    QUICKDouble DENSELJA = (QUICKDouble) LOC2(dense, LLL, JJJ, nbasis, nbasis);
                    QUICKDouble DENSELIA = (QUICKDouble) LOC2(dense, LLL, III, nbasis, nbasis);

                    QUICKDouble DENSEKIB = (QUICKDouble) LOC2(denseb, KKK, III, nbasis, nbasis);
                    QUICKDouble DENSEKJB = (QUICKDouble) LOC2(denseb, KKK, JJJ, nbasis, nbasis);
                    QUICKDouble DENSELJB = (QUICKDouble) LOC2(denseb, LLL, JJJ, nbasis, nbasis);
                    QUICKDouble DENSELIB = (QUICKDouble) LOC2(denseb, LLL, III, nbasis, nbasis);
#else
                    QUICKDouble DENSEKI = (QUICKDouble) LOC2(dense, KKK, III, nbasis, nbasis);
                    QUICKDouble DENSEKJ = (QUICKDouble) LOC2(dense, KKK, JJJ, nbasis, nbasis);
                    QUICKDouble DENSELJ = (QUICKDouble) LOC2(dense, LLL, JJJ, nbasis, nbasis);
                    QUICKDouble DENSELI = (QUICKDouble) LOC2(dense, LLL, III, nbasis, nbasis);
                    QUICKDouble DENSELK = (QUICKDouble) LOC2(dense, LLL, KKK, nbasis, nbasis);
                    QUICKDouble DENSEJI = (QUICKDouble) LOC2(dense, JJJ, III, nbasis, nbasis);
#endif

                    // ATOMIC ADD VALUE 1
                    temp = (KKK == LLL) ? DENSELK * a[k].value : 2.0 * DENSELK * a[k].value;
                    o_JI += temp;
#if defined(OSHELL)
                    ob_JI += temp;
#endif

                    // ATOMIC ADD VALUE 2
                    if (LLL != JJJ || III != KKK) {
                        temp = (III == JJJ) ? DENSEJI * a[k].value : 2.0 * DENSEJI * a[k].value;
#  if defined(USE_LEGACY_ATOMICS)
                        GPUATOMICADD(&LOC2(oULL, LLL, KKK, nbasis, nbasis), temp, OSCALE);
#  else
                        atomicAdd(&LOC2(o, LLL, KKK, nbasis, nbasis), temp);
#  endif
#if defined(OSHELL)
#  if defined(USE_LEGACY_ATOMICS)
                        GPUATOMICADD(&LOC2(obULL, LLL, KKK, nbasis, nbasis), temp, OSCALE);
#  else
                        atomicAdd(&LOC2(ob, LLL, KKK, nbasis, nbasis), temp);
#  endif
#endif
                    }

                    // ATOMIC ADD VALUE 3
#if defined(OSHELL)
                    temp = (III == KKK && III < JJJ && JJJ < LLL)
                        ? -2.0 * hyb_coeff * DENSELJA * a[k].value : -(hyb_coeff * DENSELJA * a[k].value);
                    temp2 = (III == KKK && III < JJJ && JJJ < LLL)
                        ? -2.0 * hyb_coeff * DENSELJB * a[k].value : -(hyb_coeff * DENSELJB * a[k].value);
                    o_KI += temp;
                    ob_KI += temp2;
#else
                    temp = (III == KKK && III < JJJ && JJJ < LLL)
                        ? -(hyb_coeff * DENSELJ * a[k].value) : -0.5 * hyb_coeff * DENSELJ * a[k].value;
                    o_KI += temp;
#endif

                    // ATOMIC ADD VALUE 4
                    if (KKK != LLL) {
#if defined(OSHELL)
                        temp = -(hyb_coeff * DENSEKJA * a[k].value);
                        temp2 = -(hyb_coeff * DENSEKJB * a[k].value);
#  if defined(USE_LEGACY_ATOMICS)
                        GPUATOMICADD(&LOC2(oULL, LLL, III, nbasis, nbasis), temp, OSCALE);
                        GPUATOMICADD(&LOC2(obULL, LLL, III, nbasis, nbasis), temp2, OSCALE);
#  else
                        atomicAdd(&LOC2(o, LLL, III, nbasis, nbasis), temp);
                        atomicAdd(&LOC2(ob, LLL, III, nbasis, nbasis), temp2);
#  endif
#else
                        temp = -0.5 * hyb_coeff * DENSEKJ * a[k].value;
#  if defined(USE_LEGACY_ATOMICS)
                        GPUATOMICADD(&LOC2(oULL, LLL, III, nbasis, nbasis), temp, OSCALE);
#  else
                        atomicAdd(&LOC2(o, LLL, III, nbasis, nbasis), temp);
#  endif
#endif
                    }

                    // ATOMIC ADD VALUE 5
#if defined(OSHELL)
                    temp = -(hyb_coeff * DENSELIA * a[k].value);
                    temp2 = -(hyb_coeff * DENSELIB * a[k].value);
#else
                    temp = -0.5 * hyb_coeff * DENSELI * a[k].value;
#endif
                    if ((III != JJJ && III < KKK)
                            || (III == JJJ && III == KKK && III < LLL)
                            || (III == KKK && III < JJJ && JJJ < LLL)) {
                        o_JK_MM += temp;
#if defined(OSHELL)
                        ob_JK_MM += temp2;
#endif
                    }

                    // ATOMIC ADD VALUE 5 - 2
                    if (III != JJJ && JJJ == KKK) {
                        o_JK += temp;
#if defined(OSHELL)
                        ob_JK += temp2;
#endif
                    }

                    // ATOMIC ADD VALUE 6
                    if (III != JJJ && KKK != LLL) {
#if defined(OSHELL)
                        temp = -(hyb_coeff * DENSEKIA * a[k].value);
                        temp2 = -(hyb_coeff * DENSEKIB * a[k].value);
#else
                        temp = -0.5 * hyb_coeff * DENSEKI * a[k].value;
#endif
#  if defined(USE_LEGACY_ATOMICS)
                        GPUATOMICADD(&LOC2(oULL, MAX(JJJ, LLL), MIN(JJJ, LLL), nbasis, nbasis), temp, OSCALE);
#  else
                        atomicAdd(&LOC2(o, MAX(JJJ, LLL), MIN(JJJ, LLL), nbasis, nbasis), temp);
#  endif
#if defined(OSHELL)
#  if defined(USE_LEGACY_ATOMICS)
                        GPUATOMICADD(&LOC2(obULL, MAX(JJJ, LLL), MIN(JJJ, LLL), nbasis, nbasis), temp2, OSCALE);
#  else
                        atomicAdd(&LOC2(ob, MAX(JJJ, LLL), MIN(JJJ, LLL), nbasis, nbasis), temp2);
#  endif
#endif

                        // ATOMIC ADD VALUE 6 - 2
                        if (JJJ == LLL && III != KKK) {
#  if defined(USE_LEGACY_ATOMICS)
                            GPUATOMICADD(&LOC2(oULL, LLL, JJJ, nbasis, nbasis), temp, OSCALE);
#  else
                            atomicAdd(&LOC2(o, LLL, JJJ, nbasis, nbasis), temp);
#  endif
#if defined(OSHELL)
#  if defined(USE_LEGACY_ATOMICS)
                            GPUATOMICADD(&LOC2(obULL, LLL, JJJ, nbasis, nbasis), temp2, OSCALE);
#  else
                            atomicAdd(&LOC2(ob, LLL, JJJ, nbasis, nbasis), temp2);
#  endif
#endif
                        }
                    }
                }
            }

            j = 0;
        }
    }
}
#endif


void upload_para_to_const() {
    uint8_t trans[TRANSDIM * TRANSDIM * TRANSDIM] = {};

    LOC3(trans, 0, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 0;
    LOC3(trans, 0, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 3;
    LOC3(trans, 0, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 9;
    LOC3(trans, 0, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 19;
    LOC3(trans, 0, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 34;
    LOC3(trans, 0, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 55;
    LOC3(trans, 0, 0, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 83;
    LOC3(trans, 0, 0, 7, TRANSDIM, TRANSDIM, TRANSDIM) = 119;
    LOC3(trans, 0, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 2;
    LOC3(trans, 0, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 5;
    LOC3(trans, 0, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 16;
    LOC3(trans, 0, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 31;
    LOC3(trans, 0, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 47;
    LOC3(trans, 0, 1, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 66;
    LOC3(trans, 0, 1, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 99;
    LOC3(trans, 0, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 8;
    LOC3(trans, 0, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 15;
    LOC3(trans, 0, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 22;
    LOC3(trans, 0, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 41;
    LOC3(trans, 0, 2, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 72;
    LOC3(trans, 0, 2, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 105;
    LOC3(trans, 0, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 18;
    LOC3(trans, 0, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 30;
    LOC3(trans, 0, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 42;
    LOC3(trans, 0, 3, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 78;
    LOC3(trans, 0, 3, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 111;
    LOC3(trans, 0, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 33;
    LOC3(trans, 0, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 48;
    LOC3(trans, 0, 4, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 73;
    LOC3(trans, 0, 4, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 112;
    LOC3(trans, 0, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 54;
    LOC3(trans, 0, 5, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 67;
    LOC3(trans, 0, 5, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 106;
    LOC3(trans, 0, 6, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 82;
    LOC3(trans, 0, 6, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 100;
    LOC3(trans, 0, 7, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 118;
    LOC3(trans, 1, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 1;
    LOC3(trans, 1, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 6;
    LOC3(trans, 1, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 14;
    LOC3(trans, 1, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 27;
    LOC3(trans, 1, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 49;
    LOC3(trans, 1, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 68;
    LOC3(trans, 1, 0, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 101;
    LOC3(trans, 1, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 4;
    LOC3(trans, 1, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 10;
    LOC3(trans, 1, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 25;
    LOC3(trans, 1, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 40;
    LOC3(trans, 1, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 58;
    LOC3(trans, 1, 1, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 86;
    LOC3(trans, 1, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 12;
    LOC3(trans, 1, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 24;
    LOC3(trans, 1, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 35;
    LOC3(trans, 1, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 59;
    LOC3(trans, 1, 2, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 87;
    LOC3(trans, 1, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 29;
    LOC3(trans, 1, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 39;
    LOC3(trans, 1, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 60;
    LOC3(trans, 1, 3, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 93;
    LOC3(trans, 1, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 51;
    LOC3(trans, 1, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 57;
    LOC3(trans, 1, 4, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 88;
    LOC3(trans, 1, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 70;
    LOC3(trans, 1, 5, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 85;
    LOC3(trans, 1, 6, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 103;
    LOC3(trans, 2, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 7;
    LOC3(trans, 2, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 13;
    LOC3(trans, 2, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 21;
    LOC3(trans, 2, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 43;
    LOC3(trans, 2, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 74;
    LOC3(trans, 2, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 109;
    LOC3(trans, 2, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 11;
    LOC3(trans, 2, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 23;
    LOC3(trans, 2, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 36;
    LOC3(trans, 2, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 61;
    LOC3(trans, 2, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 89;
    LOC3(trans, 2, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 20;
    LOC3(trans, 2, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 37;
    LOC3(trans, 2, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 65;
    LOC3(trans, 2, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 98;
    LOC3(trans, 2, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 45;
    LOC3(trans, 2, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 63;
    LOC3(trans, 2, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 97;
    LOC3(trans, 2, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 76;
    LOC3(trans, 2, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 91;
    LOC3(trans, 2, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 109;
    LOC3(trans, 3, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 17;
    LOC3(trans, 3, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 26;
    LOC3(trans, 3, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 44;
    LOC3(trans, 3, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 79;
    LOC3(trans, 3, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 113;
    LOC3(trans, 3, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 28;
    LOC3(trans, 3, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 38;
    LOC3(trans, 3, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 62;
    LOC3(trans, 3, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 94;
    LOC3(trans, 3, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 46;
    LOC3(trans, 3, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 64;
    LOC3(trans, 3, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 96;
    LOC3(trans, 3, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 80;
    LOC3(trans, 3, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 95;
    LOC3(trans, 3, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 115;
    LOC3(trans, 4, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 32;
    LOC3(trans, 4, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 50;
    LOC3(trans, 4, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 75;
    LOC3(trans, 4, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 114;
    LOC3(trans, 4, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 52;
    LOC3(trans, 4, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 56;
    LOC3(trans, 4, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 90;
    LOC3(trans, 4, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 77;
    LOC3(trans, 4, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 92;
    LOC3(trans, 4, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 116;
    LOC3(trans, 5, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 53;
    LOC3(trans, 5, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 69;
    LOC3(trans, 5, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 108;
    LOC3(trans, 5, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 71;
    LOC3(trans, 5, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 84;
    LOC3(trans, 5, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 110;
    LOC3(trans, 6, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 81;
    LOC3(trans, 6, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 102;
    LOC3(trans, 6, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 104;
    LOC3(trans, 7, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 117;

    gpuMemcpyToSymbol((const void *) devTrans, (const void *) trans,
            sizeof(uint8_t) * TRANSDIM * TRANSDIM * TRANSDIM);
}
